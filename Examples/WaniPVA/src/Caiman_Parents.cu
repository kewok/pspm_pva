#include "Caiman_Parents.h"
#include "Caiman.h"
#include "Caiman_mating_kernel_functors.h"

Caiman_Parents::Caiman_Parents(Caiman *species, environment *habitat) : Parents(species)
	{
	this->habitat = habitat;
	this->IRREVERSIBLE_MASS_PHENOTYPE = demeParameters->species_specific_values["IRREVERSIBLE_MASS_PHENOTYPE"];
	this->FECUNDITY_PHENOTYPE = demeParameters->species_specific_values["FECUNDITY_PHENOTYPE"];
	this->REVERSIBLE_MASS_PHENOTYPE = demeParameters->species_specific_values["REVERSIBLE_MASS_PHENOTYPE"];
	}
	
void Caiman_Parents::determine_female_parent_eligibility()
	{
	thrust::fill(will_reproduceF.begin(), will_reproduceF.begin() + size, 0);
	
	//Set up eligibility functor.
	float *f_sizes_at_maturity_ptr = raw_pointer_cast(demeParameters->get_vector_ptr("F_sizes_at_maturity"));
	parental_eligibility_functor parental_eligibility(f_sizes_at_maturity_ptr, 0);

	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(will_reproduceF.begin(), sex.begin(), deme.begin(), phenotype[IRREVERSIBLE_MASS_PHENOTYPE].begin())),
		 	thrust::make_zip_iterator(thrust::make_tuple(will_reproduceF.begin() + size, sex.begin() + size, deme.begin() + size, phenotype[IRREVERSIBLE_MASS_PHENOTYPE].begin() + size)),
			 parental_eligibility);	
	}

void Caiman_Parents::determine_male_parent_eligibility()
	{
	thrust::fill(will_reproduceM.begin(), will_reproduceM.begin() + size, 0);
	
	//Set up eligibility functor.
	float *m_sizes_at_maturity_ptr = raw_pointer_cast(demeParameters->get_vector_ptr("M_sizes_at_maturity"));
	parental_eligibility_functor parental_eligibility(m_sizes_at_maturity_ptr, 1);
		
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(will_reproduceM.begin(), sex.begin(), deme.begin(), phenotype[IRREVERSIBLE_MASS_PHENOTYPE].begin())),
			 thrust::make_zip_iterator(thrust::make_tuple(will_reproduceM.begin() + size, sex.begin() + size, deme.begin() + size, phenotype[IRREVERSIBLE_MASS_PHENOTYPE].begin() + size)),
			 parental_eligibility);	
	}

void Caiman_Parents::female_fecundity()
	{
	female_nesting_success();

	thrust::device_vector<float> egg_survivorship(size);
	thrust::gather(deme.begin(), deme.begin() + size, demeParameters->get_vector_ptr("egg_survivorship"), egg_survivorship.begin());

	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(will_reproduceF.begin(), phenotype[FECUNDITY_PHENOTYPE].begin(), egg_survivorship.begin(), probability_individual_becomes_female_parent.begin())),
			 thrust::make_zip_iterator(thrust::make_tuple(will_reproduceF.begin() + size, phenotype[FECUNDITY_PHENOTYPE].begin() + size, egg_survivorship.begin() + size, probability_individual_becomes_female_parent.begin() + size)),
			 female_fecundity_functor());

	thrust::copy(probability_individual_becomes_female_parent.begin(), probability_individual_becomes_female_parent.end(), kids_per_mom.begin());
	}

void Caiman_Parents::determine_probability_individual_becomes_female_parent()
	{
	float *f_reproductive_advantage = raw_pointer_cast(demeParameters->get_vector_ptr("F_reproductive_advantage"));
	reproductive_probability_functor F_reproductive_probability(f_reproductive_advantage, 0);

	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(will_reproduceF.begin(), deme.begin(), phenotype[FECUNDITY_PHENOTYPE].begin(), probability_individual_becomes_female_parent.begin())),
        	 thrust::make_zip_iterator(thrust::make_tuple(will_reproduceF.begin() + size, deme.begin() + size, phenotype[FECUNDITY_PHENOTYPE].begin() + size, probability_individual_becomes_female_parent.begin() + size)),
               	 F_reproductive_probability);
	}

void Caiman_Parents::determine_probability_individual_becomes_male_parent()
	{
	//Set up reproductive probability functor.
	float *m_reproductive_advantage = raw_pointer_cast(demeParameters->get_vector_ptr("M_reproductive_advantage"));

	reproductive_probability_functor M_reproductive_probability(m_reproductive_advantage, 1);

	// For now, assume male reproductive success just depends on his irreversible mass which scales with body length 

	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(will_reproduceM.begin(), deme.begin(), phenotype[IRREVERSIBLE_MASS_PHENOTYPE].begin(), probability_individual_becomes_male_parent.begin())),
        	 thrust::make_zip_iterator(thrust::make_tuple(will_reproduceM.begin() + size, deme.begin() + size,phenotype[IRREVERSIBLE_MASS_PHENOTYPE].begin() + size, probability_individual_becomes_male_parent.begin() + size)),
               	 M_reproductive_probability);
	}

void Caiman_Parents::female_nesting_success()
	{
	/* whether the female is able to nest; this effectively governs female reproductive success */
	/* NB later implementations can vary the success of the female getting a nest to depend on her relative body size etc... */
	reduce_by_key_with_zeros(deme, will_reproduceF, reproductive_females_per_deme, size, Num_Demes);

	thrust::device_vector<float> reproductive_females_per_deme_float(Num_Demes); // figure out reproductive_females

	thrust::copy(reproductive_females_per_deme.begin(), reproductive_females_per_deme.end(), reproductive_females_per_deme_float.begin());

	float *total_female_population_size_ptr = raw_pointer_cast(&reproductive_females_per_deme_float[0]);
	float *nest_ceiling_ptr = raw_pointer_cast(habitat->get_abiotic_vector_ptr("nest_ceiling"));
	int *males_per_deme_ptr = raw_pointer_cast(&reproductive_males_per_deme[0]);

	determine_female_nesting_success_functor determine_female_nesting_success(nest_ceiling_ptr, total_female_population_size_ptr, males_per_deme_ptr) ;

	thrust::device_vector<float> rand(size);
	float *rand_ptr = raw_pointer_cast(&rand[0]);
	hiprandGenerateUniform(gen, rand_ptr, size);
	
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(will_reproduceF.begin(), deme.begin(), rand.begin())), 
			thrust::make_zip_iterator(thrust::make_tuple(will_reproduceF.begin() + size, deme.begin() + size, rand.begin() + size)), 
			determine_female_nesting_success);
	}

