#include "update_Caiman.h"
#include "Caiman_Habitat.h"

void update_Caiman::update()
	{
	float *resource_1 = raw_pointer_cast(&(habitat->biotic_variables[0][0]));
	float *resource_2 = raw_pointer_cast(&(habitat->biotic_variables[1][0]));
	float *resource_3 = raw_pointer_cast(&(habitat->biotic_variables[2][0]));

	/* Determine if individuals will eat fish */
	thrust::device_vector<int> piscivory(size);
	thrust::device_vector<float> piscivory_probabilities(size);
	thrust::gather(species->deme.begin(), species->deme.begin() + size, species->demeParameters->get_vector_ptr("piscivory_probability"), piscivory_probabilities.begin());

	draw_bernoulli_different_parameters(size, piscivory_probabilities, piscivory, species->gen);

	grand_consumption_functor eat_N_grow(consumption_allometric_scalar, consumption_allometric_exponent, metabolism_allometric_scalar, metabolism_allometric_exponent, ontogenetic_niche_shift_constant, ontogenetic_niche_shift_coefficient, resource_1, resource_1_maximum, resource_2, resource_2_maximum, resource_3,  resource_3_maximum, handling_time,  functional_response_scalar, mature_maximum_condition, juvenile_maximum_condition, M_sizes_at_maturity, F_sizes_at_maturity);

	thrust::device_vector<float> resource_1_consumed(size);
	thrust::device_vector<float> resource_2_consumed(size);
	thrust::device_vector<float> resource_3_consumed(size);

	thrust::device_vector<float> rand(size);
	float *rand_ptr = raw_pointer_cast(&rand[0]);
		
	thrust::device_vector<float> temp_eaten(Number_of_Demes);
	thrust::device_vector<int> temp_demes(Number_of_Demes);
	
	// Test Force piscivory
/*	for (int i=0; i < 2; i++)
	{
	thrust::fill(piscivory.begin() + 2*i*size/(2*2), piscivory.begin() + (2*i+1)*size/(2*2), 1);
	thrust::fill(piscivory.begin() + (2*i+1)*size/(2*2), piscivory.begin() + 2*(i+1)*size/(2*2), 0);
	}
*/
	//Set up mortality functor
	calculate_mortality mortality_functor(effect_of_starvation_constant_ptr, effect_of_starvation_coefficient_ptr, maximum_survivorship_ptr, size_dependent_mortality_constant_ptr, size_dependent_mortality_coefficient_ptr);

	for (int Time_Step=0; Time_Step < intra_annual_time_steps; Time_Step++)
		{
		// Draw the random numbers for somatic growth
		hiprandGenerateUniform(species->gen, rand_ptr, size);

		thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(species->phenotype[IRREVERSIBLE_MASS_PHENOTYPE].begin(), species->phenotype[REVERSIBLE_MASS_PHENOTYPE].begin(), species->deme.begin(), species->phenotype[RESOURCE_LIMITATION_PHENOTYPE].begin(), species->sex.begin(), species->status.begin(), piscivory.begin(),  resource_1_consumed.begin(), resource_2_consumed.begin(), resource_3_consumed.begin())),
				 thrust::make_zip_iterator(thrust::make_tuple(species->phenotype[IRREVERSIBLE_MASS_PHENOTYPE].begin() + size, species->phenotype[REVERSIBLE_MASS_PHENOTYPE].begin() + size, species->deme.begin() + size, species->phenotype[RESOURCE_LIMITATION_PHENOTYPE].begin() + size,  species->sex.begin() + size, species->status.begin() + size,  piscivory.begin() + size, resource_1_consumed.begin() + size, resource_2_consumed.begin() + size, resource_3_consumed.end() + size)), 
				eat_N_grow);
	

		// Draw the random numbers for survivorship
		hiprandGenerateUniform(species->gen, rand_ptr, size);

		// Apply survivorship
		thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(species->phenotype[IRREVERSIBLE_MASS_PHENOTYPE].begin(), species->phenotype[REVERSIBLE_MASS_PHENOTYPE].begin(), species->phenotype[MORTALITY_PHENOTYPE].begin(), species->deme.begin(), species->age.begin(), species->status.begin(), rand.begin())),
				thrust::make_zip_iterator(thrust::make_tuple(species->phenotype[IRREVERSIBLE_MASS_PHENOTYPE].begin() + size, species->phenotype[REVERSIBLE_MASS_PHENOTYPE].begin() + size, species->phenotype[MORTALITY_PHENOTYPE].begin() + size, species->deme.begin() + size,  species->age.begin() + size, species->status.begin() + size, rand.begin() + size)),
				mortality_functor);
			

		// Calculate the values for the environmental variables
		hipDeviceSynchronize();
		reduce_by_key(species->deme.begin(), species->deme.begin() + size, resource_1_consumed.begin(), temp_demes.begin(), temp_eaten.begin());
		thrust::scatter(temp_eaten.begin(), temp_eaten.end(), temp_demes.begin(), habitat->effect_of_inds_on_biotic_variable[0].begin());

		reduce_by_key(species->deme.begin(), species->deme.begin() + size, resource_2_consumed.begin(), temp_demes.begin(), temp_eaten.begin());
		thrust::scatter(temp_eaten.begin(), temp_eaten.end(), temp_demes.begin(), habitat->effect_of_inds_on_biotic_variable[1].begin());

		reduce_by_key(species->deme.begin(), species->deme.begin() + size, resource_3_consumed.begin(), temp_demes.begin(), temp_eaten.begin());
		thrust::scatter(temp_eaten.begin(), temp_eaten.end(), temp_demes.begin(), habitat->effect_of_inds_on_biotic_variable[2].begin());

		habitat->update_prey();
		}

	update_vital_rates();
	}

void update_Caiman::prepare_survivorship_constants_pointers()
	{
	effect_of_starvation_coefficient_ptr = raw_pointer_cast(species->demeParameters->get_vector_ptr("effect_of_starvation_coefficient"));
	effect_of_starvation_constant_ptr = raw_pointer_cast(species->demeParameters->get_vector_ptr("effect_of_starvation_constant"));
	size_dependent_mortality_constant_ptr = raw_pointer_cast(species->demeParameters->get_vector_ptr("size_dependent_mortality_constant"));
	size_dependent_mortality_coefficient_ptr = raw_pointer_cast(species->demeParameters->get_vector_ptr("size_dependent_mortality_coefficient"));
	maximum_survivorship_ptr = raw_pointer_cast(species->demeParameters->get_vector_ptr("maximum_survivorship"));
	}

void update_Caiman::prepare_growth_constants_pointers()
	{
	/* Constants governing diet */
	ontogenetic_niche_shift_constant = raw_pointer_cast(species->demeParameters->get_vector_ptr("ontogenetic_niche_shift_constant"));
	ontogenetic_niche_shift_coefficient = raw_pointer_cast(species->demeParameters->get_vector_ptr("ontogenetic_niche_shift_coefficient"));

	resource_1_maximum = raw_pointer_cast(&(habitat->prey_array[0]->prey_maximum_abundance[0]));
	resource_2_maximum = raw_pointer_cast(&(habitat->prey_array[1]->prey_maximum_abundance[0]));
	resource_3_maximum = raw_pointer_cast(&(habitat->prey_array[2]->prey_maximum_abundance[0]));

	handling_time = raw_pointer_cast(species->demeParameters->get_vector_ptr("handling_time"));

	functional_response_scalar = raw_pointer_cast(species->demeParameters->get_vector_ptr("functional_response_numerator"));

	/* constants governing growth rates */
	mature_maximum_condition = raw_pointer_cast(species->demeParameters->get_vector_ptr("mature_maximum_condition"));
	juvenile_maximum_condition = raw_pointer_cast(species->demeParameters->get_vector_ptr("juvenile_maximum_condition"));

	consumption_allometric_scalar = raw_pointer_cast(species->demeParameters->get_vector_ptr("allometric_maximum_consumption_coefficient"));
	consumption_allometric_exponent = raw_pointer_cast(species->demeParameters->get_vector_ptr("allometric_maximum_consumption_exponent"));
	metabolism_allometric_scalar = raw_pointer_cast(species->demeParameters->get_vector_ptr("allometric_metabolic_coefficient"));
	metabolism_allometric_exponent =  raw_pointer_cast(species->demeParameters->get_vector_ptr("allometric_metabolic_exponent"));

	M_sizes_at_maturity = raw_pointer_cast(species->demeParameters->get_vector_ptr("M_sizes_at_maturity"));
	F_sizes_at_maturity = raw_pointer_cast(species->demeParameters->get_vector_ptr("F_sizes_at_maturity"));
	}

void update_Caiman::update_vital_rates()
	{
	thrust::device_vector<int> ones(size, 1);

	thrust::plus<int> add_one;
	thrust::identity<int> is_not_zero;
	thrust::transform_if(species->age.begin(), species->age.begin() + size, ones.begin(), species->status.begin(), species->age.begin(), add_one, is_not_zero);
	update_fecundity();	
	}

void update_Caiman::update_fecundity()
	{
	//Set up reproductive contribution functor.
	float *juvenile_maximum_conditions_ptr = raw_pointer_cast(species->demeParameters->get_vector_ptr("juvenile_maximum_condition"));

	update_female_fecundity_functor fecundity(juvenile_maximum_conditions_ptr);

	thrust::device_vector<int> indices(size);
	thrust::sequence(indices.begin(), indices.begin() + size);

	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(species->sex.begin(), species->deme.begin(), species->phenotype[IRREVERSIBLE_MASS_PHENOTYPE].begin(), species->phenotype[REVERSIBLE_MASS_PHENOTYPE].begin(), species->phenotype[EGGSIZE_PHENOTYPE].begin(), species->phenotype[FECUNDITY_PHENOTYPE].begin(), indices.begin())),
			 thrust::make_zip_iterator(thrust::make_tuple(species->sex.begin() + size, species->deme.begin() + size, species->phenotype[IRREVERSIBLE_MASS_PHENOTYPE].begin() + size, species->phenotype[REVERSIBLE_MASS_PHENOTYPE].begin() + size, species->phenotype[EGGSIZE_PHENOTYPE].begin() + size, species->phenotype[FECUNDITY_PHENOTYPE].begin() + size, indices.end())),
			 fecundity);
	}
