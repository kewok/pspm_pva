#include "Caiman_genotype_phenotype_maps.h"

GenotypePhenotypeMap *GenotypePhenotypeMap::create_genotype_phenotype_map(inds *species, int phenotype_index, int index_case, int num_kids)
	{
	if (phenotype_index == species->demeParameters->species_specific_values["FECUNDITY_PHENOTYPE"]) 
		{
		return new fecundity_phenotype(species, phenotype_index, index_case, num_kids);
		}

	if (phenotype_index == species->demeParameters->species_specific_values["MORTALITY_PHENOTYPE"])
		{
		return new mortality_phenotype(species, phenotype_index, index_case, num_kids);
		}

	if (phenotype_index == species->demeParameters->species_specific_values["IRREVERSIBLE_MASS_PHENOTYPE"])
		{
		return new irreversible_mass_at_birth(species, phenotype_index, index_case, num_kids);
		}

	if (phenotype_index == species->demeParameters->species_specific_values["REVERSIBLE_MASS_PHENOTYPE"])
		{
		return new reversible_mass_at_birth(species, phenotype_index, index_case, num_kids);
		}

	if (phenotype_index == species->demeParameters->species_specific_values["EGGSIZE_PHENOTYPE"])
		{
		return new egg_size(species, phenotype_index, index_case, num_kids);
		}

	if (phenotype_index == species->demeParameters->species_specific_values["INBREEDING_PHENOTYPE"])
		{
		return new inbreeding_at_birth(species, phenotype_index, index_case, num_kids);
		}
	}


