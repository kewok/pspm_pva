#include "Caiman_Simulator.h"
#include <util/footimer2.h>

#define BURNIN_PERIOD 100

Caiman_Simulator::Caiman_Simulator() : Simulation()
	{
	initpop = 500*demes;
	maxpop = 50000*demes;
	
	nspecies = 1;
	initialize_classes();
	}

void Caiman_Simulator::initialize_classes()
	{
	hipSetDevice(0);
	hipDeviceSynchronize();

	habitat = new Caiman_Habitat(seed, num_biotic_variables, num_abiotic_variables, demes, intra_step_time_steps);
	hipDeviceSynchronize();

	array = new inds_stochastic *[nspecies];
	hipDeviceSynchronize();
	int species_ID = 0;

	array[0] = new Caiman(initpop, maxpop, seed, demes, species_ID);

	stats_eggsize = new Statistics(demes, "summary_statistics_eggsize.txt", "useless_histograms.txt");
	stats_fecundity = new Statistics(demes, "summary_statistics_fecundity.txt", "useless_histograms2.txt");

	//array[0]-> exportCsv("initial_data.csv");

	demographics = new DemographicStatistics(demes, "demographic_statistics.txt", "age_distribution.txt");

	preyfile.open("prey_sizes.txt");
	}

void Caiman_Simulator::run()
	{
	footimer2 timer, timerAll;
	timerAll.start();

	for (int t=0; t < nsteps; t++)
		{
		for (int i=0; i < nspecies; i++)
			{
			array[i]->addKids(habitat);
			array[i]->update(array, habitat);
			array[i]->removeDead();

			hipDeviceSynchronize();

			demographics->calculate_deme_sizes(array[i]);
			demographics->record_deme_sizes();
			}

		if (t > BURNIN_PERIOD)
			{
			habitat->update();
			}

		for (int k=0; k < demes; k++)
			{
			preyfile << habitat->prey_array[0]->prey_abundance[k] << " " << habitat->prey_array[1]->prey_abundance[k] << " " << habitat->prey_array[2]->prey_abundance[k] << " ";
			}
		preyfile << std::endl;
		}
	timerAll.stop();
	std::cout << "Total "; 
	timerAll.printTime();
	}

Caiman_Simulator::~Caiman_Simulator()
	{
	/* cleanup */
	for (int i=0; i < nspecies; i++)
		{
		delete array[i];
		}

	delete[] array;
	delete habitat;

	delete stats_fecundity;
	delete stats_eggsize;
	delete demographics;
	}
