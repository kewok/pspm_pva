#include "prey_class.h"
#include <math/random_variables_functions.h>

#include <fstream>
#include <fstream>
#include <iostream>
#include <thrust/functional.h>


typedef thrust::device_vector<float>::iterator floatIter_t;
typedef thrust::device_vector<int>::iterator intIter_t;

prey::prey(int num_demes, int prey_index, int seed_val, int num_time_steps) : Num_Demes(num_demes), Prey_Index(prey_index), seed (seed_val),  Number_of_Timesteps (num_time_steps)
	{
	prey_abundance.resize(Num_Demes);
	prey_maximum_abundance.resize(Num_Demes);
	prey_assimilation_efficiency.resize(Num_Demes);

	prey_carrying_capacity.resize(Num_Demes);
	prey_unconstrained_growth_rates.resize(Num_Demes);

	prey_growth_rate_noise_stddev.resize(Num_Demes);

	//Initialize hiprand generator.
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, seed);

	prey_abundance.resize(Num_Demes);

	noise_updated = 0;

	//Read in deme-specific data for each prey type	
	preyParameters = new PreyDemeSpecificData("environment_config.txt", prey_index);
	}

void prey::specify_prey_properties_by_deme()
	{
	thrust::copy(preyParameters->get_vector_ptr("prey_density_dependence"), preyParameters->get_vector_ptr("prey_density_dependence") + Num_Demes, prey_carrying_capacity.begin());

	thrust::copy(preyParameters->get_vector_ptr("prey_unconstrained_growth_rates"), preyParameters->get_vector_ptr("prey_unconstrained_growth_rates") + Num_Demes, prey_unconstrained_growth_rates.begin());

	thrust::copy(preyParameters->get_vector_ptr("prey_assimilation_efficiency"), preyParameters->get_vector_ptr("prey_assimilation_efficiency") + Num_Demes, prey_assimilation_efficiency.begin());

	thrust::copy(preyParameters->get_vector_ptr("prey_maximum_abundance"), preyParameters->get_vector_ptr("prey_maximum_abundance") + Num_Demes, prey_maximum_abundance.begin());

	// initialize the prey abundances.
	thrust::copy(prey_maximum_abundance.begin(), prey_maximum_abundance.begin() + Num_Demes, prey_abundance.begin());

	// Initialize noise to zero
	thrust::fill(prey_growth_rate_noise_stddev.begin(), prey_growth_rate_noise_stddev.begin() + Num_Demes, 0.0);
	}

void prey::update_prey_abundance(thrust::device_vector<float> &effect_of_individuals_on_prey)
	{
	// Add random noise to the prey abundance
	thrust::device_vector<float> stochastic_component(Num_Demes);
	thrust::device_vector<float> stochastic_growth(Num_Demes);
	thrust::device_vector <float> zeros(Num_Demes);
	thrust::fill(zeros.begin(), zeros.begin() + Num_Demes, 0);

	draw_gaussian_different_parameters(Num_Demes, zeros, prey_growth_rate_noise_stddev, stochastic_component, gen);

	thrust::transform(prey_unconstrained_growth_rates.begin(), prey_unconstrained_growth_rates.begin() + Num_Demes, stochastic_component.begin(), stochastic_component.begin(), thrust::multiplies<float>());
	thrust::transform(prey_unconstrained_growth_rates.begin(), prey_unconstrained_growth_rates.begin() + Num_Demes, stochastic_component.begin(), stochastic_growth.begin(), thrust::plus<float>());	

	// typedef for clarity
	typedef thrust::tuple<floatIter_t, floatIter_t, floatIter_t, floatIter_t> tuple_t;
	typedef thrust::zip_iterator<tuple_t> zipIter_t;

	tuple_t start = thrust::make_tuple(stochastic_growth.begin(), prey_carrying_capacity.begin(), effect_of_individuals_on_prey.begin(), prey_abundance.begin());
	tuple_t end = thrust::make_tuple(stochastic_growth.begin()  + Num_Demes, prey_carrying_capacity.begin()  + Num_Demes, effect_of_individuals_on_prey.begin() + Num_Demes, prey_abundance.begin() + Num_Demes);

 	//Create zip iterators.
	zipIter_t zstart = thrust::make_zip_iterator(start);
	zipIter_t zend = thrust::make_zip_iterator(end);

	thrust::for_each(zstart, zend,  update_prey());

	hipDeviceSynchronize();
	// Make sure you have no negative prey
	thrust::replace_if(prey_abundance.begin(), prey_abundance.begin() + Num_Demes, is_less_than_zero_f(), 0.0);
	}

void prey::update_prey_density_dependence()
	{
	thrust::for_each(
		thrust::make_zip_iterator(thrust::make_tuple(prey_carrying_capacity.begin(), preyParameters->get_vector_ptr("prey_density_dependence_change"))),
		thrust::make_zip_iterator(thrust::make_tuple(prey_carrying_capacity.begin() + Num_Demes, preyParameters->get_vector_ptr("prey_density_dependence_change") + Num_Demes)),
		update_density_dependence());
	}

void prey::update_prey_growth_noise()
	{
	if (noise_updated == 0)
		{
		thrust::copy(preyParameters->get_vector_ptr("prey_growth_rate_noise_stddev"), preyParameters->get_vector_ptr("prey_growth_rate_noise_stddev") + Num_Demes, prey_growth_rate_noise_stddev.begin());
		noise_updated = 1;
		}
	}
