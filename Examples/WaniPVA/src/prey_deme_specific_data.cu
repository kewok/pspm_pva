#include "prey_deme_specific_data.h"

PreyDemeSpecificData::PreyDemeSpecificData(const char *filename, int prey_index)
	{
	read_in_prey_parameters(filename, prey_index);
	specify_parameter_index();
	}

void PreyDemeSpecificData::specify_parameter_index()
	{
	for (int i=0; i < parameter_names.size(); i++)
		parameter_index[parameter_names[i]] = i;
	}

void PreyDemeSpecificData::read_in_prey_parameters(const char *filename, int prey_index)
	{
	Config cfg;
	try
		{
		cfg.readFile(filename);
		}
	catch(const FileIOException &fioex)
		{
		std::cerr << "No prey config file." << std::endl;
		}
	catch(const ParseException &pex)
		{
		std::cerr << "While setting up prey, your " << pex.getFile() << " file is incorrectly specified. Make sure you check on or about line: " << pex.getLine() << " - " << pex.getError() << std::endl;
		}
	try
		{
		Number_of_Parameters = cfg.lookup("number_of_parameters");
		}
	catch(const SettingNotFoundException &nfex)
		{
		std::cerr << "No 'number_of_parameters' setting in prey configuration file." << std::endl;
		}

	try
		{
		const Setting& root = cfg.getRoot();
		parameter_names.resize(Number_of_Parameters);

		if (root["parameter_names"].getLength() != Number_of_Parameters)
			std::cout << "Length of parameter names differs from number of parameters\n" << std::endl;

		for (int i=0; i < root["parameter_names"].getLength(); i++)
				{
				parameter_names[i] = root["parameter_names"][i].c_str();
				}

		const Setting &deme_specifications = root["prey_initialization"][prey_index];

		Number_of_Demes = deme_specifications.getLength();

		deme_wide_parameters = new thrust::device_vector<float>[Number_of_Parameters];

		for (int i=0; i < Number_of_Parameters; i++)
			deme_wide_parameters[i].resize(Number_of_Demes);

		for (int i=0; i < root["parameter_names"].getLength(); i++)
				{
				for (int j=0; j < Number_of_Demes; j++)
					{
					const Setting &deme_values = deme_specifications[j];
					float val = 0;
					deme_values.lookupValue(parameter_names[i], val);
					deme_wide_parameters[i][j] = val;
					}
				}
		}
	
	catch(const SettingNotFoundException &nfex)
		{
		std::cerr << "Your prey configuration file does not work." << std::endl;
		}
	}

thrust::device_ptr<float> PreyDemeSpecificData::get_vector_ptr(const char *parameter_name)
	{
	return(&deme_wide_parameters[parameter_index[parameter_name]][0]);
	}
