#include <gtest/gtest.h>
#include <gmock/gmock.h>
#include <ConfigFile.h>
#include <iomanip>
#include "Caiman.h"
#include "Caiman_Habitat.h"

class Test_Mass_Growth : public ::testing::Test{
	protected:
		virtual void setup() {
		}
		virtual void TearDown() {
		}
};

TEST_F(Test_Mass_Growth, update_caiman) 
	{
	ConfigFile config("Simulation.conf");
	int demes = config.read<int>( "ndemes" );
	int intra_step_time_steps = config.read<int>( "intra_step_time_steps" );

	int seed = config.read<int>("random_seed");

	int num_biotic_variables = config.read<int>( "num_biotic_variables" );
	int num_abiotic_variables = config.read<int>( "num_abiotic_variables" );

	inds_stochastic **TEST_Caiman;
	TEST_Caiman = new inds_stochastic *[0];
	Caiman_Habitat *habitat;

	int initpop = 10*demes;
	int maxpop = 10*demes;
	int species_ID = 0;

	habitat = new Caiman_Habitat(seed, num_biotic_variables, num_abiotic_variables, demes, intra_step_time_steps);
	TEST_Caiman[0] = new Caiman(initpop, maxpop, seed, demes, species_ID);
	hipDeviceSynchronize();

//	Prey Abundance
/*	habitat->prey_array[0]->prey_abundance[0] = 175000;
	habitat->prey_array[0]->prey_abundance[1] = 17500000;

	habitat->prey_array[1]->prey_abundance[0] = 20000;
	habitat->prey_array[1]->prey_abundance[1] = 2000000;

	habitat->prey_array[2]->prey_abundance[0] = 10070;
	habitat->prey_array[2]->prey_abundance[1] = 1007000;
*/

	TEST_Caiman[0]->phenotype[3][0] = 38.2;
	TEST_Caiman[0]->phenotype[4][0] = 76.4;
	TEST_Caiman[0]->phenotype[3][1] = 5832;
	TEST_Caiman[0]->phenotype[4][1] = 11664;
	TEST_Caiman[0]->phenotype[3][2] = 9664;
	TEST_Caiman[0]->phenotype[4][2] = 19328;
	TEST_Caiman[0]->phenotype[3][3] = 5832;
	TEST_Caiman[0]->phenotype[4][3] = 16329;
	TEST_Caiman[0]->phenotype[3][4] = 9664;
	TEST_Caiman[0]->phenotype[4][4] = 27059;
	TEST_Caiman[0]->phenotype[3][5] = 38.2;
	TEST_Caiman[0]->phenotype[4][5] = 76.4;
	TEST_Caiman[0]->phenotype[3][6] = 5832;
	TEST_Caiman[0]->phenotype[4][6] = 11664;
	TEST_Caiman[0]->phenotype[3][7] = 9664;
	TEST_Caiman[0]->phenotype[4][7] = 19328;
	TEST_Caiman[0]->phenotype[3][8] = 5832;
	TEST_Caiman[0]->phenotype[4][8] = 16329;
	TEST_Caiman[0]->phenotype[3][9] = 9664;
	TEST_Caiman[0]->phenotype[4][9] = 27059;
	TEST_Caiman[0]->phenotype[3][10] = 38.2;
	TEST_Caiman[0]->phenotype[4][10] = 76.4;
	TEST_Caiman[0]->phenotype[3][11] = 5832;
	TEST_Caiman[0]->phenotype[4][11] = 11664;
	TEST_Caiman[0]->phenotype[3][12] = 9664;
	TEST_Caiman[0]->phenotype[4][12] = 19328;
	TEST_Caiman[0]->phenotype[3][13] = 5832;
	TEST_Caiman[0]->phenotype[4][13] = 16329;
	TEST_Caiman[0]->phenotype[3][14] = 9664;
	TEST_Caiman[0]->phenotype[4][14] = 27059;
	TEST_Caiman[0]->phenotype[3][15] = 38.2;
	TEST_Caiman[0]->phenotype[4][15] = 76.4;
	TEST_Caiman[0]->phenotype[3][16] = 5832;
	TEST_Caiman[0]->phenotype[4][16] = 11664;
	TEST_Caiman[0]->phenotype[3][17] = 9664;
	TEST_Caiman[0]->phenotype[4][17] = 19328;
	TEST_Caiman[0]->phenotype[3][18] = 5832;
	TEST_Caiman[0]->phenotype[4][18] = 16329;
	TEST_Caiman[0]->phenotype[3][19] = 9664;
	TEST_Caiman[0]->phenotype[4][19] = 27059;


	TEST_Caiman[0] -> update(TEST_Caiman, habitat);
	TEST_Caiman[0] -> addKids();

/*	habitat->update();

	for (int i=0; i < habitat->nbiotic_vars; i++)
		{
		for (int j=0; j < demes; j++)
			{
			std::cout<< "Prey type " << i << " count deme " << j << " " << habitat->prey_array[i]->prey_abundance[j] <<std::endl;
			}
		}
*/
//	Define vector
	thrust::host_vector<float> expected_answer_rev_mas(maxpop);
	thrust::host_vector<float> expected_answer_irrev_mas(maxpop);
	thrust::host_vector<float> expected_answer_fecund(maxpop);
	thrust::host_vector<float> expected_resource1_levels(demes);
	thrust::host_vector<float> expected_resource2_levels(demes);
	thrust::host_vector<float> expected_resource3_levels(demes);


//	Manually Add Values.
	expected_answer_rev_mas[0]=74.3035781035;
	expected_answer_rev_mas[1]=11551.43467;
	expected_answer_rev_mas[2]=19161.5235706;
	expected_answer_rev_mas[3]=16193.7933693;
	expected_answer_rev_mas[4]=19328;
	expected_answer_rev_mas[5]=74.3035781035;
	expected_answer_rev_mas[6]=11551.43467;
	expected_answer_rev_mas[7]=19161.5235706;
	expected_answer_rev_mas[8]=11664;
	expected_answer_rev_mas[9]=26859.1334045;
	expected_answer_rev_mas[10]=81.0144017632;
	expected_answer_rev_mas[11]=11715.5393843;
	expected_answer_rev_mas[12]=19371.8564762;
	expected_answer_rev_mas[13]=16373.7527234;
	expected_answer_rev_mas[14]=19393.6477187;
	expected_answer_rev_mas[15]=81.0144017632;
	expected_answer_rev_mas[16]=11701.1013816;
	expected_answer_rev_mas[17]=19397.4565977;
	expected_answer_rev_mas[18]=11720.3739486;
	expected_answer_rev_mas[19]=27123.8852862;


	expected_answer_irrev_mas[0]=38.2;
	expected_answer_irrev_mas[1]=5832;
	expected_answer_irrev_mas[2]=9664;
	expected_answer_irrev_mas[3]=5832;
	expected_answer_irrev_mas[4]=9664;
	expected_answer_irrev_mas[5]=38.2;
	expected_answer_irrev_mas[6]=5832;
	expected_answer_irrev_mas[7]=9664;
	expected_answer_irrev_mas[8]=5832;
	expected_answer_irrev_mas[9]=9664;
	expected_answer_irrev_mas[10]=40.5072008816;
	expected_answer_irrev_mas[11]=5857.76969213;
	expected_answer_irrev_mas[12]=9685.92823808;
	expected_answer_irrev_mas[13]=5871.15593533;
	expected_answer_irrev_mas[14]=9696.82385937;
	expected_answer_irrev_mas[15]=40.5072008816;
	expected_answer_irrev_mas[16]=5850.5506908;
	expected_answer_irrev_mas[17]=9685.92823808;
	expected_answer_irrev_mas[18]=5860.18697429;
	expected_answer_irrev_mas[19]=9696.82385937;

	expected_answer_fecund[0]=0;
	expected_answer_fecund[1]=0;
	expected_answer_fecund[2]=0;
	expected_answer_fecund[3]=0;
	expected_answer_fecund[4]=65;
	expected_answer_fecund[5]=0;
	expected_answer_fecund[6]=0;
	expected_answer_fecund[7]=0;
	expected_answer_fecund[8]=39;
	expected_answer_fecund[9]=0;
	expected_answer_fecund[10]=0;
	expected_answer_fecund[11]=0;
	expected_answer_fecund[12]=0;
	expected_answer_fecund[13]=0;
	expected_answer_fecund[14]=67;
	expected_answer_fecund[15]=0;
	expected_answer_fecund[16]=0;
	expected_answer_fecund[17]=0;
	expected_answer_fecund[18]=40;
	expected_answer_fecund[19]=0;

	expected_resource1_levels[0]=760664.898231621
;	expected_resource1_levels[1]=17497875.4037023
;

	expected_resource2_levels[0]=58822.0923797183
;	expected_resource2_levels[1]=1999985.0825332
;

	expected_resource3_levels[0]=11898.8507923049
;	expected_resource3_levels[1]=993659.115866533
;

	thrust::host_vector<float> host_answer_irrev_mas = TEST_Caiman[0]->phenotype[3];
	thrust::host_vector<float> host_answer_rev_mas = TEST_Caiman[0]->phenotype[4];
	thrust::host_vector<float> host_resource1_levels = habitat->prey_array[0]->prey_abundance;
	thrust::host_vector<float> host_resource2_levels = habitat->prey_array[1]->prey_abundance;
	thrust::host_vector<float> host_resource3_levels = habitat->prey_array[2]->prey_abundance;
	thrust::host_vector<float> host_answer_fecund = TEST_Caiman[0]->phenotype[0];

	EXPECT_THAT(host_answer_rev_mas, ::testing::ContainerEq(expected_answer_rev_mas));
	EXPECT_THAT(host_answer_irrev_mas, ::testing::ContainerEq(expected_answer_irrev_mas));
	EXPECT_THAT(host_answer_fecund, ::testing::ContainerEq(expected_answer_fecund));
	for (int i=0; i < host_resource1_levels.size(); i++)
		{
		EXPECT_FLOAT_EQ(host_resource1_levels[i], expected_resource1_levels[i]); //::testing::ContainerEq(expected_resource1_levels));
		EXPECT_FLOAT_EQ(host_resource2_levels[i], expected_resource2_levels[i]); //::testing::ContainerEq(expected_resource2_levels));
		EXPECT_FLOAT_EQ(host_resource3_levels[i], expected_resource3_levels[i]); //::testing::ContainerEq(expected_resource3_levels));
		}
	}

