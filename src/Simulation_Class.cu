#include "Simulation_Class.h"

using namespace libconfig;

Simulation::Simulation()
	{
	read_simulation_settings();
	extern const char *gitversion;
	std::ofstream file;
	file.open("associated_git_commit.txt"); 
	file << gitversion << std::endl;
	file.close();
	}
	
void Simulation::read_simulation_settings()
	{
	Config cfg;

	try
		{
		cfg.readFile("Simulation.conf");
		}
	catch(const FileIOException &fioex)
		{
		std::cerr << "No Simulation.conf file." << std::endl;
		}
	catch(const ParseException &pex)
		{
		std::cerr << "Your " << pex.getFile() << " file is incorrectly specified. Make sure you check on or about line: " << pex.getLine() << " - " << pex.getError() << std::endl;
		}
	try
		{
		nsteps = cfg.lookup("n_timesteps");
		}
 	catch(const SettingNotFoundException &nfex)
		{
		std::cerr << "No 'n_timesteps' setting in configuration file." << std::endl;
		}
	try
		{
		demes = cfg.lookup("ndemes");
		}
 	catch(const SettingNotFoundException &nfex)
		{
		std::cerr << "No 'ndemes' setting in configuration file." << std::endl;
		}
	try
		{
		num_biotic_variables = cfg.lookup("num_biotic_variables");
		}
 	catch(const SettingNotFoundException &nfex)
		{
		std::cerr << "No 'num_biotic_variables' setting in configuration file." << std::endl;
		}
	try
		{
		num_abiotic_variables = cfg.lookup("num_abiotic_variables");
		}
 	catch(const SettingNotFoundException &nfex)
		{
		std::cerr << "No 'num_abiotic_variables' setting in configuration file." << std::endl;
		}
	// Optional entries
	try
		{
		intra_step_time_steps = cfg.lookup("intra_step_time_steps");
		}
	catch(const SettingNotFoundException &nfex)
		{
		// Ignore.
		}
	try
		{
		seed = cfg.lookup("random_seed");
		}
	catch(const SettingNotFoundException &nfex)
		{
		// Ignore.
		}
	// Individual csv recording times
	try
		{
		const Setting& root = cfg.getRoot();
		const Setting &output_csv_steps = root["output_csv_steps"];
		for (int i=0; i < output_csv_steps.getLength(); i++)
			{
			steps_to_output_individuals_csv.push_back(output_csv_steps[i]);
			std::cout << steps_to_output_individuals_csv[i] << std::endl;
			}
		}
	catch(const SettingNotFoundException &nfex)
		{
		// Ignore.
		}
	}

Simulation::~Simulation()
	{
	}
