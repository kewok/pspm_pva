#include <environ/environment.h>

#include <hiprand.h>
#include <fstream>
#include <iostream>
#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/scatter.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

environment::environment(int seed_val, int num_biotic_variables, int num_abiotic_variables, int num_demes) : seed(seed_val), ndemes(num_demes), nbiotic_vars(num_biotic_variables), nabiotic_vars(num_abiotic_variables)
	{
	//Allocate biotic data vectors.
	biotic_variables = new thrust::device_vector<float>[nbiotic_vars];
	effect_of_inds_on_biotic_variable = new thrust::device_vector<float>[nbiotic_vars];

	// Resize to accord with number of demes
	for (int i = 0 ; i < nbiotic_vars ; i++) 
		{
		biotic_variables[i].resize(ndemes);
		effect_of_inds_on_biotic_variable[i].resize(ndemes);
		}

		// Initialize the first part of the feedback process 
	for (int i = 0 ; i < nbiotic_vars ; i++) 
		{
		thrust::fill(effect_of_inds_on_biotic_variable[i].begin(), effect_of_inds_on_biotic_variable[i].begin() + ndemes, 0);
		}

	// Allocate abiotic data vectors
	abiotic_variables = new thrust::device_vector<float>[nabiotic_vars];

	for (int i=0; i < nabiotic_vars; i++)
		{
		abiotic_variables[i].resize(ndemes);
		}

	//Initialize hiprand generator.
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, seed);
	}

void environment::initialize_abiotic_variables(const char *filename)
        {
        Config cfg;
        try
                {
                cfg.readFile(filename);
                }
        catch(const FileIOException &fioex)
                {
                std::cerr << "No environmental config file available" << std::endl;
                }
	catch(const ParseException &pex)
		{
		std::cerr << "Your " << pex.getFile() << " file is incorrectly specified. Make sure you check on or about line: " << pex.getLine() << " - " << pex.getError() << std::endl;
		}


        const Setting &root = cfg.getRoot();

        const Setting &abiotic_variable_specification = root["abiotic_variable_names"];

        int lenVal = abiotic_variable_specification.getLength();

        abiotic_variable_names.resize(lenVal);

        for (int i=0; i < lenVal; i++)
                {
                abiotic_variable_names[i] = abiotic_variable_specification[i].c_str();
                }

        // Read in the values for each abiotic variable

        const Setting &abiotic_variable_values = root["abiotic_variables"] ;

        abiotic_variables = new thrust::device_vector<float>[nabiotic_vars];

        for (int i=0; i < nabiotic_vars; i++)
                {
                abiotic_variables[i].resize(ndemes);
                }

        for (int i=0; i < nabiotic_vars; i++)
                {
                for (int j=0; j < ndemes; j++)
                        {
                        const Setting &deme_values = abiotic_variable_values[j];
                        float val = 0;
                        deme_values.lookupValue(abiotic_variable_names[i], val);
                        abiotic_variables[i][j] = val;
                        }
                }

        // specify the indices associated with each abiotic variable's name:

        for (int i=0; i < nabiotic_vars; i++)
                {
                abiotic_variable_indices[abiotic_variable_names[i]] = i;
                }
        }

thrust::device_ptr<float> environment::get_abiotic_vector_ptr(const char *abiotic_variable_name)
	{
	return(&abiotic_variables[abiotic_variable_indices[abiotic_variable_name]][0]);
	}


environment::~environment()
	{
	delete[] biotic_variables;
	delete[] effect_of_inds_on_biotic_variable;
	}


