#include <util/Sample_With_Replacement.h>
#include <util/amplify.h>
#include <util/thrust_functors.h>

void Sample_With_Replacement::sample()
	{
	/*************
	//
	// 1. For each sampling event, generate a vector "individuals_sampling" that repeats each index i in (0,1,2,... , |x|) n_i times.
	//
	**************/

	amplify(sampling_input->list_of_individuals_potentially_conducting_sampling, sampling_input->number_of_other_individuals_sampled, sampling_individuals_indices);

	/*************
	//
	// 2. Perform the stratified sampling by deme
	//
	**************/

		/***********
		//
		// 2.a Generate a vector u, where the jth element of u is the cumulative number of individuals subject to sampling in deme j
		//
		***********/

	// The individuals subject to sampling have to fall within the ranges of the total number of individuals in the sampler's deme. Also make sure Num_Demes is initialized; if is set to zero you will get a memory error.


	thrust::device_vector<int> cumulative_sampleable_individuals_by_deme( sampling_input->Num_Demes );

	thrust::inclusive_scan( sampling_input->sampleable_individuals_per_deme.begin(), sampling_input-> sampleable_individuals_per_deme.end() , cumulative_sampleable_individuals_by_deme.begin() );	

	int *cumulative_others_by_deme_ptr = raw_pointer_cast(&cumulative_sampleable_individuals_by_deme[0]);

	thrust::device_vector<int> deme_of_samplers;

	amplify(sampling_input->deme_affiliation_of_sampling_individuals, sampling_input->number_of_other_individuals_sampled, deme_of_samplers);

		/***********
		//
		// 2.b. For each individual i that is doing the sampling, draw a random float z[i] between 0 and 1
		//
		***********/

	thrust::device_vector<float> sampled_floats( sampling_individuals_indices.size() ); 
	float *curand_samples_ptr = raw_pointer_cast( &sampled_floats[0] );
	hiprandGenerateUniform(gen, curand_samples_ptr, sampling_individuals_indices.size());

		/***********
		//
		// 2.c. Let that random number fall between u[j-1], u[j] where j is the deme of the ith individual sampler.
		//
		***********/

	// sample total_others_sampled other individuals at random
	thrust::device_vector<int> individuals_sampled( sampling_individuals_indices.size() ); 

	// Use the functor reassign to change the random integer to fall within the range of individuals in the deme
	reassign_functor reassign(cumulative_others_by_deme_ptr);

	thrust::for_each( thrust::make_zip_iterator(thrust::make_tuple(
					sampled_floats.begin(),
					deme_of_samplers.begin(),
					individuals_sampled.begin()
					)),
			  thrust::make_zip_iterator(thrust::make_tuple(
					sampled_floats.end(),
					deme_of_samplers.end(),
					individuals_sampled.end())),
				reassign
				);

	/*************
	//
	// 3. Map Z[i] to the vector w to produce a vector W, the list of individuals that get sampled for each sampling event  (the gather operation)
	//
	**************/

	// Specify the index of the sampled individual whose index in individuals_sampled corresponds to:	
	// Make sure you use the index from the overall deme to which the sampled individuals belong. 

	sampled_individuals_indices.resize( sampling_individuals_indices.size() );
	thrust::gather( individuals_sampled.begin(), individuals_sampled.end(), sampling_input-> list_of_individuals_potentially_subject_to_sampling.begin(),  sampled_individuals_indices.begin());
	}

