#include <util/sampling_event.h>
#include <util/Sample_With_Replacement.h>
#include <util/Sample_without_Replacement_1Pass.h>
#include <util/Sample_without_Replacement_Test.h>
#include <util/Sample_without_Replacement_byDeme.h>
// #include "Sample_without_Replacement.h" // For now, this isn't quite ready yet

// TODO: right now have no way of returning NULL if error?

SamplingEvent *SamplingEvent::create_SamplingEvent(SamplingInput *sampling_input, hiprandGenerator_t gen)
	{
	if (sampling_input->sampling_scheme == "SAMPLE_WITH_REPLACEMENT")
		{
		return new Sample_With_Replacement(sampling_input, gen);
		}

	if (sampling_input->sampling_scheme == "SAMPLE_WITHOUT_REPLACEMENT_1Pass")
		{
		return new Sample_without_Replacement_1Pass(sampling_input, gen);
		}

	if (sampling_input->sampling_scheme == "SAMPLE_WITHOUT_REPLACEMENT")
		{
		return new Sample_without_Replacement_Test(sampling_input, gen);
		}

	if (sampling_input->sampling_scheme == "SAMPLE_WITHOUT_REPLACEMENT_BYDEME")
		{
		return new Sample_without_Replacement_byDeme(sampling_input, gen);
		}
 /*
// not ready 
  if (sampling_input->sampling_scheme == 3)
    return new Sample_without_Replacement(sampling_input, gen);

*/
	}


