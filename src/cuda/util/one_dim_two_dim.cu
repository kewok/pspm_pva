#include "hip/hip_runtime.h"
#include <util/one_dim_two_dim.h>

struct one_dim_two_dim_functor
	{
/* 
*For a one dimensional vector made by converting a two-dimensional * vector into a 1D vector, find the corresponding values from the original 2-D vector. 
If you have a vector of int values
A = [A0, A1, A2, A3, ..., An]

and another vector of int values 

B = [B0, B1, B2, B3, ..., Bm] 

then form a third vector

C = [C0, C1, ...., Cm, C(m+1), ...,C(nxm)]

where C0 corresponds to [A0,B0], C1 corresponds to [A0,B1], C2 to [A0,B2], ...,Cm to [A0,Bm], C(m+1) to [A1,B0],..., C(nxm) to [An,Bm].

Conversely, if you have a vector

C = [C0, C1, ...., Cm, C(m+1), ...,C(nxm)]

then one_dim_two_dim works such that for each C_j, return the pair [Ai,Bk] that correspond to C_j, e.g., one_dim_two_dim(C(m+1), length(A), length(B))=[A1,B0].

Note that all of these are o(1) operations per variable element.
*/

	int length_vector_2;

	one_dim_two_dim_functor(int len_vector_2) : length_vector_2(len_vector_2)
	{};
	/* 
		Elements in the tuple.

		----------------------
		0: input value from the single-dimensioned array
		1: corresponding output value for the first array
		2: corresponding output value for the second array
	*/ 
	template <typename tuple>
	__host__ __device__
	void operator()(tuple t) {
		float one_dim_val = (float) thrust::get<0>(t);
		float len_vec_2 = (float) length_vector_2;
		thrust::get<1>(t) = floorf(one_dim_val/len_vec_2);
		thrust::get<2>(t) = thrust::get<0>(t) - thrust::get<1>(t)*length_vector_2;
		}
	};

void one_dim_two_dim(thrust::device_vector<int> &vector1_values,
	     thrust::device_vector<int> &vector2_values,
	     thrust::device_vector<int> &new_vector,
	     thrust::device_vector<int> &values_for_vector_1,
	     thrust::device_vector<int> &values_for_vector_2)
	{

	/* If you have a vector of int values
	*A = [A0, A1, A2, A3, ..., An]
	*and another vector of int values 
	*B = [B0, B1, B2, B3, ..., Bm] 
	*then form a third vector
	*C = [C0, C1, ...., Cm, C(m+1), ...,C(nxm)]
	*where C0 corresponds to [A0,B0], C1 corresponds to [A0,B1], C2 to [A0,B2], ...,Cm to [A0,Bm], C(m+1) to [A1,B0],..., C(nxm) to [An,Bm].
	*Conversely, if you have a vector
	*C = [C0, C1, ...., Cm, C(m+1), ...,C(nxm)]
	*then one_dim_two_dim works such that for each C_j, return the pair [Ai,Bk] that correspond to C_j, e.g., one_dim_two_dim(C(m+1), length(A), length(B))=[A1,B0].
	*Note that all of these are o(1) operations per variable element.
	*/
	int length_new_vector = new_vector.size();
	int length_vector_2 = vector2_values.size();

	thrust::device_vector<int> indices_vector_1 ( length_new_vector );
	thrust::device_vector<int> indices_vector_2 ( length_new_vector );

	one_dim_two_dim_functor get_two_dim(length_vector_2);
		
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(new_vector.begin(), indices_vector_1.begin(), indices_vector_2.begin())), 	
			 thrust::make_zip_iterator(thrust::make_tuple(new_vector.end(), indices_vector_1.end(), indices_vector_2.end())),
			 get_two_dim);

	thrust::gather(indices_vector_1.begin(), indices_vector_1.end(), vector1_values.begin() , values_for_vector_1.begin());
	thrust::gather(indices_vector_2.begin(), indices_vector_2.end(), vector2_values.begin() , values_for_vector_2.begin());
	}

