#include <util/sampling_input.h>
#include <species/inds.h>
//#include "species_specific_sampling_input.h"
#include <math/histogram.h>

// The operations in prepare_sampling_input() are also shared across species and contexts

void SamplingInput::setup_sampling_individuals_demes(int num_demes, thrust::device_vector<int> &sampling_individuals_demes)
	{
	Num_Demes = num_demes;
	deme_affiliation_of_sampling_individuals.resize(sampling_individuals_demes.size());
	thrust::copy(sampling_individuals_demes.begin(), sampling_individuals_demes.end(), 	deme_affiliation_of_sampling_individuals.begin());	
	}

void SamplingInput::setup_sampleable_individuals_per_deme(thrust::device_vector<int> &num_sampleable_individuals_in_demes)
	{
	sampleable_individuals_per_deme.resize(Num_Demes);
	thrust::copy(num_sampleable_individuals_in_demes.begin(), num_sampleable_individuals_in_demes.end(), sampleable_individuals_per_deme.begin());

	// In demes with zero sampleable individuals, excise from the list of individuals potentially conducting sampling and their demes 
	int smallest_deme = *(thrust::min_element(sampleable_individuals_per_deme.begin(), sampleable_individuals_per_deme.end()));

	if (smallest_deme == 0)
		{
		thrust::device_vector<int> sampleable_individuals_in_sampling_individuals_deme(deme_affiliation_of_sampling_individuals.size());
		thrust::gather(deme_affiliation_of_sampling_individuals.begin(), deme_affiliation_of_sampling_individuals.end(), sampleable_individuals_per_deme.begin(), sampleable_individuals_in_sampling_individuals_deme.begin());

		list_of_individuals_potentially_conducting_sampling.erase(thrust::remove_if(list_of_individuals_potentially_conducting_sampling.begin(), list_of_individuals_potentially_conducting_sampling.end(), sampleable_individuals_in_sampling_individuals_deme.begin(), unary_less_equal<int>(0)), list_of_individuals_potentially_conducting_sampling.end());

		if (deme_affiliation_of_sampling_individuals.size() > list_of_individuals_potentially_conducting_sampling.size())
			{
			deme_affiliation_of_sampling_individuals.erase(thrust::remove_if(deme_affiliation_of_sampling_individuals.begin(), deme_affiliation_of_sampling_individuals.end(), sampleable_individuals_in_sampling_individuals_deme.begin(), unary_less_equal<int>(0)), deme_affiliation_of_sampling_individuals.end());
			}

		if (number_of_other_individuals_sampled.size() >  list_of_individuals_potentially_conducting_sampling.size())
			{
			number_of_other_individuals_sampled.erase(thrust::remove_if(number_of_other_individuals_sampled.begin(), number_of_other_individuals_sampled.end(), sampleable_individuals_in_sampling_individuals_deme.begin(), unary_less_equal<int>(0)), number_of_other_individuals_sampled.end());
			}
		hipDeviceSynchronize();
		}
	}

#if 0
void SamplingInput::prepare_sampling_input()
	{
	specify_individuals_conducting_sampling();
	specify_individuals_being_sampled();
	set_parameters();
	determine_number_of_individuals_sampled();
	}


// Methods to determine how many individuals each individual samples
void SamplingInput::determine_number_of_individuals_to_be_sampled_poisson() 
	{

	int number_of_individuals_doing_the_sampling = list_of_individuals_potentially_conducting_sampling.size();

	// For each individual, based on their deme, identify the expected number of other individuals they are likely to sample
	thrust::device_vector<float> mean_numbers_of_others_sampled( number_of_individuals_doing_the_sampling );
	
	thrust::gather(population_affiliation_of_sampling_individuals.begin(), 
		       population_affiliation_of_sampling_individuals.end(), 
		       species[focal_species_ID]->subpopParameters->get_vector_ptr("mean_number_of_others_sampled"),
		       mean_numbers_of_others_sampled.begin());

	// draw a poisson, where mean_numbers_of_others_sampled is the mean.
	draw_poisson_different_parameters(number_of_individuals_doing_the_sampling, 
					  mean_numbers_of_others_sampled, 
					  number_of_other_individuals_sampled,
					  species[focal_species_ID]->gen);

	// Make sure you sample no more than there are individuals in the other species deme by:
		// 1. Figuring out the demes of individuals subject to being sampled
	thrust::device_vector<int> maximum_sampleable_individuals( number_of_individuals_doing_the_sampling );

	thrust::device_vector<int> target_species_deme_values( list_of_individuals_potentially_subject_to_sampling.size() );

	thrust::gather(list_of_individuals_potentially_subject_to_sampling.begin(),
		       list_of_individuals_potentially_subject_to_sampling.end(),
		       species[target_species_ID]->pop.begin(),
		       target_species_deme_values.begin());
	
		// 2. Count the number of times each deme is represented
	thrust::device_vector<int> possible_deme_values( species[target_species_ID]->Num_Subpopulations );
	thrust::sequence(possible_deme_values.begin(), possible_deme_values.end());

	calculate_histogram(target_species_deme_values, sampleable_individuals_per_deme, species[target_species_ID]->Num_Subpopulations);

	thrust::gather(population_affiliation_of_sampling_individuals.begin(),
		       population_affiliation_of_sampling_individuals.end(),
		       sampleable_individuals_per_deme.begin(),
		       maximum_sampleable_individuals.begin());

		// 3. Apply correction for small deme size: cannot sample more individuals than there are individuals in the deme.
	thrust::transform( number_of_other_individuals_sampled.begin(),  number_of_other_individuals_sampled.end(), maximum_sampleable_individuals.begin(),  number_of_other_individuals_sampled.begin(), thrust::minimum<int>());
	}


// Method to set the same number of others sampled for all individuals in a deme
void SamplingInput::determine_number_of_individuals_to_be_sampled_fixed() 
	{
	// Assume all individuals sample the same number of individuals in the other species.
	thrust::device_vector<int> maximum_sampleable_individuals( number_of_individuals_in_sampling_species );
	thrust::device_vector<int> target_species_deme_values( list_of_individuals_potentially_subject_to_sampling.size() );

	thrust::gather(list_of_individuals_potentially_subject_to_sampling.begin(),
		       list_of_individuals_potentially_subject_to_sampling.end(),
		       species[target_species_ID]->pop.begin(),
		       target_species_deme_values.begin());
	
	thrust::device_vector<int> possible_deme_values( species[target_species_ID]->Num_Subpopulations );
	thrust::sequence(possible_deme_values.begin(), possible_deme_values.end());

	calculate_histogram(target_species_deme_values, sampleable_individuals_per_deme, species[target_species_ID]->Num_Subpopulations);

	thrust::gather(population_affiliation_of_sampling_individuals.begin(),
		       population_affiliation_of_sampling_individuals.end(),
		       sampleable_individuals_per_deme.begin(),
		       maximum_sampleable_individuals.begin());

	// specify the number of other individuals sampled, which we assume is set deterministically
	thrust::gather(population_affiliation_of_sampling_individuals.begin(), 
		       population_affiliation_of_sampling_individuals.end(), 
		       species[focal_species_ID]->subpopParameters->get_vector_ptr("mean_number_of_others_sampled"),
		       number_of_other_individuals_sampled.begin());
	

	// Apply correction for small deme size: cannot sample more individuals than there are individuals in the deme.
	thrust::transform( number_of_other_individuals_sampled.begin(),  number_of_other_individuals_sampled.end(), maximum_sampleable_individuals.begin(),  number_of_other_individuals_sampled.begin(), thrust::minimum<int>());
	}
#endif
