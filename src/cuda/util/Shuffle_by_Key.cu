#include <util/shuffle_by_key.h>

void shuffle_by_key(thrust::device_vector<int> &keys, thrust::device_vector<int> &values, hiprandGenerator_t gen, bool shuffle_keys)
	{
	int max_number_of_duplicates = 1;
	thrust::device_vector<double> unique_uniform_rvs(keys.size());

	while(max_number_of_duplicates > 0)
		{
		// Note this routine suffers from the same interminable problem outlined in Sample_without_Replacement_Test if there are identical uniform rvs. Test simulations suggest the scenarios that will cause you troubles is unlikely to arise frequently in practice with 10^7 double variates
		double *rand_ptr = raw_pointer_cast(&unique_uniform_rvs[0]);
		hiprandGenerateUniformDouble(gen, rand_ptr, keys.size());

		thrust::device_vector<double> temp_keys(keys.size());

		thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(keys.begin(), temp_keys.begin())),
				 thrust::make_zip_iterator(thrust::make_tuple(keys.end(), temp_keys.end())), 
				 int_to_double());

		thrust::transform(temp_keys.begin(), temp_keys.end(), unique_uniform_rvs.begin(), unique_uniform_rvs.begin(), thrust::plus<double>());
		hipDeviceSynchronize();
		/* Test for any duplicates */
		thrust::device_vector<double> sorted_rand(keys.size());

		thrust::copy(unique_uniform_rvs.begin(), unique_uniform_rvs.end(), sorted_rand.begin());
		thrust::sort(sorted_rand.begin(), sorted_rand.end());
		// If there are duplicates, this line of code should remove them from the sorted_rand vector.
		sorted_rand.erase(thrust::unique(sorted_rand.begin(), sorted_rand.end()), sorted_rand.end());
		// Consequently, the sorted_rand vector will be shorter than the original uniform RV vector, necessitating a new set of uniform RVs be drawn:
		max_number_of_duplicates = unique_uniform_rvs.size() - sorted_rand.size();	
		// Provided there were no duplicates
		if (max_number_of_duplicates == 0)
			{
			if (shuffle_keys)
				{
				thrust::device_vector<double> unique_uniform_copy(keys.size());
				thrust::copy(unique_uniform_rvs.begin(), unique_uniform_rvs.end(), unique_uniform_copy.begin());
				thrust::stable_sort_by_key(unique_uniform_rvs.begin(), unique_uniform_rvs.end(), values.begin());
				thrust::stable_sort_by_key(unique_uniform_copy.begin(), unique_uniform_copy.end(), keys.begin());
				}
			else
				{
				thrust::stable_sort_by_key(unique_uniform_rvs.begin(), unique_uniform_rvs.end(), values.begin());
				}
			}
		}
	}
