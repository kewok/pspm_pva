#include "hip/hip_runtime.h"
#include <util/Sample_without_Replacement_1Pass.h>
#include <util/amplify.h>
#include <util/thrust_functors.h>
#include <util/Sample_With_Replacement.h>
#include <util/remove_duplicate_pairs.h>

void Sample_without_Replacement_1Pass::sample()
	{
	// Reset sampling_input to allow sampling with replacement for one round:
	sampling_input->sampling_scheme = "SAMPLE_WITH_REPLACEMENT";

	class SamplingEvent *sample_others;
	sample_others = sample_others->create_SamplingEvent(sampling_input, gen);
	sample_others->sample();

	sampling_individuals_indices.resize(sample_others->sampling_individuals_indices.size());
	sampled_individuals_indices.resize(sample_others->sampled_individuals_indices.size());

	thrust::copy(sample_others->sampling_individuals_indices.begin(), sample_others->sampling_individuals_indices.end(), sampling_individuals_indices.begin());

	thrust::copy(sample_others->sampled_individuals_indices.begin(), sample_others->sampled_individuals_indices.end(), sampled_individuals_indices.begin());

	// Remove duplicates
	remove_duplicate_pairs(sampling_individuals_indices, sampled_individuals_indices);
	}

