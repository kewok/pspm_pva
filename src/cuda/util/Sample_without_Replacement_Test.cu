#include <util/Sample_without_Replacement_Test.h>
#include <util/thrust_functors.h>
#include <util/amplify.h>

#include <thrust/unique.h>

// This class draws a single sample, sampled without replacement, for each individual potentially conducting the sampling.

void Sample_without_Replacement_Test::sample()
	{
	 sampled_individuals_indices.resize(number_of_individuals_subject_to_sampling);
	thrust::copy(sampling_input->list_of_individuals_potentially_subject_to_sampling.begin(), sampling_input->list_of_individuals_potentially_subject_to_sampling.end(),  sampled_individuals_indices.begin());

	shuffle_sampled_individuals_at_random();

	amplify(sampling_input->list_of_individuals_potentially_conducting_sampling, sampling_input->number_of_other_individuals_sampled, sampling_individuals_indices);

	sampled_individuals_indices.erase(sampled_individuals_indices.begin() + sampling_individuals_indices.size(), sampled_individuals_indices.end());
	}

void Sample_without_Replacement_Test::shuffle_sampled_individuals_at_random()
	{
	unique_uniform_rvs.resize(number_of_individuals_subject_to_sampling);

	int max_number_of_duplicates = 1;

	while(max_number_of_duplicates > 0)
		{
		// This algorithm can be interminable if number_of_individuals_subject_to_sampling is huge (not clear what magic number is; testing seems to suggest it still handles about 70 million random variates fine, which is at the memory limit of the GPU)
		double *rand_ptr = raw_pointer_cast(&unique_uniform_rvs[0]);
		hiprandGenerateUniformDouble(gen, rand_ptr, number_of_individuals_subject_to_sampling); 

		thrust::device_vector<double> sorted_rand(number_of_individuals_subject_to_sampling);
		thrust::copy(unique_uniform_rvs.begin(), unique_uniform_rvs.end(), sorted_rand.begin());
		thrust::sort(sorted_rand.begin(), sorted_rand.end());
		// If there are duplicates, this line of code should remove them from the sorted_rand vector.
		sorted_rand.erase(thrust::unique(sorted_rand.begin(), sorted_rand.end()), sorted_rand.end());
		// Consequently, the sorted_rand vector will be shorter than the original uniform RV vector, necessitating a new set of uniform RVs be drawn:
		max_number_of_duplicates = unique_uniform_rvs.size() - sorted_rand.size();
		
		if (max_number_of_duplicates == 0)
			{
			thrust::stable_sort_by_key(unique_uniform_rvs.begin(), unique_uniform_rvs.end(),  sampled_individuals_indices.begin());
			}
		}
	}
