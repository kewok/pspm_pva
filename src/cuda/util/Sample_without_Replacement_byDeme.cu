#include <math/histogram.h>
#include <util/Sample_without_Replacement_byDeme.h>

void Sample_without_Replacement_byDeme::sample()
	{
	int max_number_of_duplicates = 1;

	/* For each of the individuals conducting the sampling, identify the index of the shuffled individual to which they would be assigned */
	int *cumulative_demewise_sum_species_conducting_sampling = raw_pointer_cast(&cumulative_sampling_individuals_by_deme[0]);
	int *cumulative_demewise_sum_species_subject_to_sampling = raw_pointer_cast(&cumulative_sampleable_individuals_by_deme[0]);

	thrust::device_vector<int> sampling_individuals(sampling_individuals_indices.size());
	thrust::sequence(sampling_individuals.begin(), sampling_individuals.end());
	// Identify the index of the individual to be sampled, stratified by deme
	specify_index_to_sample idx_to_sample(cumulative_demewise_sum_species_conducting_sampling, cumulative_demewise_sum_species_subject_to_sampling);

	thrust::for_each(thrust::make_zip_iterator(
			 	thrust::make_tuple(sampling_individuals.begin(), sampling_input->deme_affiliation_of_sampling_individuals.begin(), index_to_sample.begin())), 
			 thrust::make_zip_iterator(
			 	thrust::make_tuple(sampling_individuals.end(), sampling_input->deme_affiliation_of_sampling_individuals.end(), index_to_sample.end())), idx_to_sample);

	hipDeviceSynchronize();

	while(max_number_of_duplicates > 0)
		{
		// Note this routine suffers from the same interminable problem outlined in Sample_without_Replacement_Test if there are identical uniform rvs. Test simulations suggest the scenarios that will cause you troubles is unlikely to arise frequently in practice with 10^7 double variates
		double *rand_ptr = raw_pointer_cast(&unique_uniform_rvs[0]);
		hiprandGenerateUniformDouble(gen, rand_ptr, sampling_input->list_of_individuals_potentially_subject_to_sampling.size());

		thrust::device_vector<double> temp_demes(demes_of_individuals_subject_to_sampling.size());

		thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(demes_of_individuals_subject_to_sampling.begin(), temp_demes.begin())),
				 thrust::make_zip_iterator(thrust::make_tuple(demes_of_individuals_subject_to_sampling.end(), temp_demes.end())), 
				 int_to_double());

		thrust::transform(temp_demes.begin(), temp_demes.end(), unique_uniform_rvs.begin(), unique_uniform_rvs.begin(), thrust::plus<double>());
		hipDeviceSynchronize();
		/* Test for any duplicates */
		thrust::device_vector<double> sorted_rand(sampling_input->list_of_individuals_potentially_subject_to_sampling.size());

		thrust::copy(unique_uniform_rvs.begin(), unique_uniform_rvs.end(), sorted_rand.begin());
		thrust::sort(sorted_rand.begin(), sorted_rand.end());
		// If there are duplicates, this line of code should remove them from the sorted_rand vector.
		sorted_rand.erase(thrust::unique(sorted_rand.begin(), sorted_rand.end()), sorted_rand.end());
		// Consequently, the sorted_rand vector will be shorter than the original uniform RV vector, necessitating a new set of uniform RVs be drawn:
		max_number_of_duplicates = unique_uniform_rvs.size() - sorted_rand.size();	
		// Provided there were no duplicates
		if (max_number_of_duplicates == 0)
			{
			thrust::device_vector<int> temp_sampled_individuals_indices(sampling_input->list_of_individuals_potentially_subject_to_sampling.size());
			thrust::copy(sampling_input-> list_of_individuals_potentially_subject_to_sampling.begin(), sampling_input-> list_of_individuals_potentially_subject_to_sampling.end(), temp_sampled_individuals_indices.begin());
			thrust::stable_sort_by_key(unique_uniform_rvs.begin(), unique_uniform_rvs.end(), temp_sampled_individuals_indices.begin());

			thrust::gather(index_to_sample.begin(), index_to_sample.end(), temp_sampled_individuals_indices.begin(),  sampled_individuals_indices.begin());
			sampled_individuals_indices.erase(sampled_individuals_indices.begin() + sampling_individuals_indices.size(), sampled_individuals_indices.end());
			}
		}
	}

void Sample_without_Replacement_byDeme::setup_demes(thrust::device_vector<int> &demes_of_individuals_sampled)
	{
	thrust::copy(demes_of_individuals_sampled.begin(), demes_of_individuals_sampled.end(), demes_of_individuals_subject_to_sampling.begin());
	calculate_histogram(demes_of_individuals_sampled, sampling_input->sampleable_individuals_per_deme, sampling_input->Num_Demes);

	thrust::exclusive_scan(sampling_input->sampleable_individuals_per_deme.begin(), sampling_input->sampleable_individuals_per_deme.end(), cumulative_sampleable_individuals_by_deme.begin());
	
	calculate_histogram(sampling_input->deme_affiliation_of_sampling_individuals, number_of_sampling_individuals_by_deme, sampling_input->Num_Demes);
	thrust::exclusive_scan(number_of_sampling_individuals_by_deme.begin(),  number_of_sampling_individuals_by_deme.end(), cumulative_sampling_individuals_by_deme.begin());
	}
