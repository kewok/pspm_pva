#include <math/mating_thrust_prob_table_demes.h>
#include <util/footimer2.h>
#include <util/thrust_functors.h>

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/distance.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/gather.h>
#include <thrust/unique.h>

/*

Here is the idea with this implementation of thrust_prob_table

when we have keys [1,1,2,3,4,4,4] and probabilities [0.2,0.3,0.1,0.6,0.1,0.25,0.3], do an inclusive scan on the vector of probabilities, i.e., 
prob' = [0.20, 0.50, 0.60, 1.20, 1.30, 1.55, 1.85]
Do a histogram of [1, 1, 2, 3, 4, 4, 4] to get vector hist = [2, 1, 1, 3] (this is presumably done ahead of time, and the histogram is fed in as an argument to determine_key_offsets)
take an inclusive scan of hist, hist' = [2, 3, 4, 7]. These are your offsets.

Then for each uniform random variable u, if u is associated with key[i], u' = prob'[hist'[key[i]-1] ] + u*(prob'[hist'[key[i]] ]  - prob'[hist'[key[i]-1] ] )
Then draw based on u.
*/

// make sure the kid's uniform random number falls in the interval corresponding to their current population
struct adjust_randoms_functor
	{
	float *popRanges_in_cumulative;

	adjust_randoms_functor(float *pop_range) : popRanges_in_cumulative(pop_range)
	{};
	
	/*
		Elements in the tuple.
		----------------------
		0: the kid's random number
		1: the kid's population
	*/
	template <typename tuple>
	__host__ __device__
	void operator()(tuple t) {
			int this_kids_pop = thrust::get<1>(t);
			if (this_kids_pop != 0)
				{
				float interval = popRanges_in_cumulative[this_kids_pop] - popRanges_in_cumulative[this_kids_pop - 1];
				thrust::get<0>(t) = thrust::get<0>(t)*interval +  popRanges_in_cumulative[this_kids_pop-1];
				}

	 // For the case of the kids in population zero

			else
				{
				float interval = popRanges_in_cumulative[this_kids_pop];
				thrust::get<0>(t) = thrust::get<0>(t)*interval;
				}
		}
	};



struct adjust_randoms_functor_double
	{
	double *popRanges_in_cumulative;

	adjust_randoms_functor_double(double *pop_range) : popRanges_in_cumulative(pop_range)
	{};
	
	/*
		Elements in the tuple.
		----------------------
		0: the kid's random number
		1: the kid's population
	*/
	template <typename tuple>
	__host__ __device__
	void operator()(tuple t) {
			int this_kids_pop = thrust::get<1>(t);
			if (this_kids_pop != 0)
				{
				double interval = popRanges_in_cumulative[this_kids_pop] - popRanges_in_cumulative[this_kids_pop - 1];
				thrust::get<0>(t) = thrust::get<0>(t)*interval +  popRanges_in_cumulative[this_kids_pop-1];
				}

	 // For the case of the kids in population zero

			else
				{
				double interval = popRanges_in_cumulative[this_kids_pop];
				thrust::get<0>(t) = thrust::get<0>(t)*interval;
				}
		}
	};


void mating_ThrustProbTable_demes::determine_key_offsets(int number_of_key_types, thrust::device_vector<int> &key_histogram_vector )
	{
	thrust::device_vector<int> temp_offsets( number_of_key_types ); 
	thrust::inclusive_scan(key_histogram_vector.begin(), key_histogram_vector.end(), temp_offsets.begin());

	// Need to subtract one from the offsets (as offsets are based on counts, but we need to match index which starts at zero)
	key_offsets.resize( number_of_key_types );	
	thrust::transform_if(temp_offsets.begin(), temp_offsets.begin() + number_of_key_types, key_offsets.begin(), unary_minus<unsigned int>(1), unary_greater<unsigned int>(0));
	}

void mating_ThrustProbTable_demes::adjust_randoms(thrust::device_vector<float>::iterator uniform_begin, thrust::device_vector<float>::iterator uniform_end,
thrust::device_vector<int>::iterator inds_demes_begin, thrust::device_vector<int>::iterator inds_demes_end)
	{
	// Note the number of populations.

	int n = thrust::distance(key_offsets.begin(), key_offsets.end());

	// Figure out the bounds for each subpopulation in terms of their values in the cumulative probability table
	thrust::device_vector<float> bounds(n);

	thrust::gather(key_offsets.begin(), key_offsets.end(),cumulative_prob.begin(),bounds.begin());
	
	thrust::device_vector<int> temp(n);
	thrust::copy(key_offsets.begin(), key_offsets.end(), temp.begin());

	float *cumul_prob_bounds = raw_pointer_cast(&bounds[0]);
	// Instantiate the random number adjuster
	adjust_randoms_functor adjuster(cumul_prob_bounds);

	// Adjust the random numbers to fall inside the correct interval
	thrust::for_each( thrust::make_zip_iterator(thrust::make_tuple(
				uniform_begin, 
				inds_demes_begin
				)),
			thrust::make_zip_iterator(thrust::make_tuple(
				uniform_end,
				inds_demes_end					
				)),
				adjuster
				);
	}


void mating_ThrustProbTable_demes::adjust_randoms(thrust::device_vector<double>::iterator uniform_begin, thrust::device_vector<double>::iterator uniform_end,
thrust::device_vector<int>::iterator inds_demes_begin, thrust::device_vector<int>::iterator inds_demes_end)
	{
	// Note the number of populations.

	int n = thrust::distance(key_offsets.begin(), key_offsets.end());

	// Figure out the bounds for each subpopulation in terms of their values in the cumulative probability table
	thrust::device_vector<double> bounds(n);

	thrust::gather(key_offsets.begin(), key_offsets.end(),cumulative_prob_double.begin(),bounds.begin());
	
	thrust::device_vector<int> temp(n);
	thrust::copy(key_offsets.begin(), key_offsets.end(), temp.begin());

	double *cumul_prob_bounds = raw_pointer_cast(&bounds[0]);
	// Instantiate the random number adjuster
	adjust_randoms_functor_double adjuster(cumul_prob_bounds);

	// Adjust the random numbers to fall inside the correct interval
	thrust::for_each( thrust::make_zip_iterator(thrust::make_tuple(
				uniform_begin, 
				inds_demes_begin
				)),
			thrust::make_zip_iterator(thrust::make_tuple(
				uniform_end,
				inds_demes_end					
				)),
				adjuster
				);
	}

