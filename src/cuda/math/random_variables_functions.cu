#include <math/random_variables_functions.h>
#include <math/thrust_probabilities.h>

#include <hiprand.h>
#include <fstream>
#include <iostream>

void prime_random_number_generator(hiprandGenerator_t gen, int seed)
	{
/*
*
* Not clear why having this be in random_variables_functions does not work when prime_random_number_generator gets called from inds_stochastic's initialization. it is likely an issue of code not linking correctly...
*
*/
	int size = 100;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, seed);

	//hiprand declarations
	thrust::device_vector<float> rand(size);
	float *rand_ptr = raw_pointer_cast(&rand[0]);
	hiprandGenerateUniform(gen, rand_ptr, size); // priming up the random number generator takes some time, get it done early.
	rand.clear();
	}

void draw_gaussian(int samples_needed, float mean, float stddev, thrust::device_vector<float> &random_variates, hiprandGenerator_t gen)
	{
/*
*
* a wrapper for drawing gaussian random variates - hiprandGenerateNormal() behaves when interacting with thrust vectors. This routine is slower than hiprandGenerateNormal(), so waiting for workable thrust generator to come out.
*
*/
	thrust::device_vector<float> rand1(samples_needed); 
	float *rand_ptr1 = raw_pointer_cast(&rand1[0]);
	thrust::device_vector<float> rand2(samples_needed); 
	float *rand_ptr2 = raw_pointer_cast(&rand2[0]);

	hiprandGenerateUniform(gen, rand_ptr1, samples_needed);
	hiprandGenerateUniform(gen, rand_ptr2, samples_needed);

	normal_rv gen_normal(mean, stddev);
		
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(rand1.begin(),rand2.begin(),random_variates.begin())),
			thrust::make_zip_iterator(thrust::make_tuple(rand1.begin() + samples_needed,rand2.begin() + samples_needed,random_variates.begin()  + samples_needed)), gen_normal);
	}

void draw_discrete_gaussian(int samples_needed, float mean, float stddev, thrust::device_vector<int> &random_variates, hiprandGenerator_t gen)
	{
	thrust::device_vector<float> rand1(samples_needed); 
	float *rand_ptr1 = raw_pointer_cast(&rand1[0]);
	thrust::device_vector<float> rand2(samples_needed); 
	float *rand_ptr2 = raw_pointer_cast(&rand2[0]);

	hiprandGenerateUniform(gen, rand_ptr1, samples_needed);
	hiprandGenerateUniform(gen, rand_ptr2, samples_needed);

	discrete_normal_rv gen_discrete_normal(mean, stddev);
		
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(rand1.begin(),rand2.begin(),random_variates.begin())),
			thrust::make_zip_iterator(thrust::make_tuple(rand1.begin() + samples_needed,rand2.begin() + samples_needed,random_variates.begin()  + samples_needed)), gen_discrete_normal);
	}




void draw_poisson(int samples_needed, float lambda, thrust::device_vector<int> &random_variates, hiprandGenerator_t gen)
	{
/*
*
* NB: This is something that apparently CUDA 5 also implements. Eventually we should switch to using cuda's native poisson and gaussian simulator
*
*/
	thrust::device_vector<float> rand1(samples_needed); 
	float *rand_ptr1 = raw_pointer_cast(&rand1[0]);
	
	hiprandGenerateUniform(gen, rand_ptr1, samples_needed);

	thrust::transform(rand1.begin(), rand1.end(), random_variates.begin(), poisson_rv(lambda));
	}

void draw_gaussian_different_parameters(int samples_needed, thrust::device_vector<float> &Means, thrust::device_vector<float> &Standard_deviations, thrust::device_vector<float> &random_variates, hiprandGenerator_t gen)
	{
	thrust::device_vector<float> rand1(samples_needed); 
	float *rand_ptr1 = raw_pointer_cast(&rand1[0]);
	thrust::device_vector<float> rand2(samples_needed); 
	float *rand_ptr2 = raw_pointer_cast(&rand2[0]);

	hiprandGenerateUniform(gen, rand_ptr1, samples_needed);
	hiprandGenerateUniform(gen, rand_ptr2, samples_needed);

	thrust::for_each( thrust::make_zip_iterator(thrust::make_tuple(rand1.begin(),rand2.begin(),Means.begin(),Standard_deviations.begin(), random_variates.begin() )),
		thrust::make_zip_iterator(thrust::make_tuple(rand1.begin() + samples_needed,rand2.begin() + samples_needed, Means.begin() + samples_needed, Standard_deviations.begin() + samples_needed, random_variates.begin() + samples_needed)),normal_rv_different_parameters());		
	}


void draw_gaussian_different_parameters(int samples_needed, float mean_value, thrust::device_vector<float> &Standard_deviations, thrust::device_vector<float> &random_variates, hiprandGenerator_t gen)
	{
	thrust::device_vector<float> Means(samples_needed);
	thrust::fill(Means.begin(), Means.end(), mean_value);

	thrust::device_vector<float> rand1(samples_needed); 
	float *rand_ptr1 = raw_pointer_cast(&rand1[0]);
	thrust::device_vector<float> rand2(samples_needed); 
	float *rand_ptr2 = raw_pointer_cast(&rand2[0]);

	hiprandGenerateUniform(gen, rand_ptr1, samples_needed);
	hiprandGenerateUniform(gen, rand_ptr2, samples_needed);

	thrust::for_each( thrust::make_zip_iterator(thrust::make_tuple(rand1.begin(),rand2.begin(),Means.begin(),Standard_deviations.begin(), random_variates.begin() )),
		thrust::make_zip_iterator(thrust::make_tuple(rand1.begin() + samples_needed,rand2.begin() + samples_needed, Means.begin() + samples_needed, Standard_deviations.begin() + samples_needed, random_variates.begin() + samples_needed)),normal_rv_different_parameters());		
	}

void draw_discrete_gaussian_different_parameters(int samples_needed, thrust::device_vector<float> &Means, thrust::device_vector<float> &Standard_deviations, thrust::device_vector<int> &random_variates, hiprandGenerator_t gen)
	{
	thrust::device_vector<float> rand1(samples_needed); 
	float *rand_ptr1 = raw_pointer_cast(&rand1[0]);
	thrust::device_vector<float> rand2(samples_needed); 
	float *rand_ptr2 = raw_pointer_cast(&rand2[0]);

	hiprandGenerateUniform(gen, rand_ptr1, samples_needed);
	hiprandGenerateUniform(gen, rand_ptr2, samples_needed);

	thrust::for_each( thrust::make_zip_iterator(thrust::make_tuple(rand1.begin(),rand2.begin(),Means.begin(),Standard_deviations.begin(), random_variates.begin() )),
		thrust::make_zip_iterator(thrust::make_tuple(rand1.begin() + samples_needed,rand2.begin() + samples_needed, Means.begin() + samples_needed, Standard_deviations.begin() + samples_needed, random_variates.begin() + samples_needed)),discrete_normal_rv_different_parameters());		
	}

void  draw_poisson_different_parameters(int samples_needed, thrust::device_vector<float> &lambdas, thrust::device_vector<int> &random_variates, hiprandGenerator_t gen)
	{
	thrust::device_vector<float> rand1(samples_needed); 
	float *rand_ptr1 = raw_pointer_cast(&rand1[0]);

	hiprandGenerateUniform(gen, rand_ptr1, samples_needed);
	
	thrust::for_each( thrust::make_zip_iterator(thrust::make_tuple(rand1.begin(),lambdas.begin(), random_variates.begin() )),
		thrust::make_zip_iterator(thrust::make_tuple(rand1.begin() + samples_needed, lambdas.begin() + samples_needed, random_variates.begin() + samples_needed)), poisson_rv_different_parameters());		
	}


void draw_bernoulli(int samples_needed, float probability, thrust::device_vector<int> &random_variates, hiprandGenerator_t gen)
	{
	thrust::device_vector<float> rand1(samples_needed); 
	float *rand_ptr1 = raw_pointer_cast(&rand1[0]);
	hiprandGenerateUniform(gen, rand_ptr1, samples_needed);

	thrust::fill(random_variates.begin(), random_variates.end(), 0);

	thrust::transform(rand1.begin(), rand1.end(), random_variates.begin(), bernoulli_rv(probability));
	}

void draw_bernoulli(int samples_needed, float probability, thrust::device_vector<float> &random_variates, hiprandGenerator_t gen)
	{
	thrust::device_vector<float> rand1(samples_needed); 
	float *rand_ptr1 = raw_pointer_cast(&rand1[0]);
	hiprandGenerateUniform(gen, rand_ptr1, samples_needed);

	thrust::fill(random_variates.begin(), random_variates.end(), 0);

	thrust::device_vector<int> temporary_random_variates(samples_needed);

	thrust::transform(rand1.begin(), rand1.end(), temporary_random_variates.begin(), bernoulli_rv(probability));

	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(temporary_random_variates.begin(), random_variates.begin())),
		 thrust::make_zip_iterator(thrust::make_tuple(temporary_random_variates.begin()+samples_needed, random_variates.begin()+samples_needed)),
		 int_to_float());
	}

void draw_bernoulli_different_parameters(int samples_needed, thrust::device_vector<float> &probabilities, thrust::device_vector<int> &random_variates, hiprandGenerator_t gen)
	{
/*
*
* Draw a sequence of samples_needed Bernoulli random variables where the probability of success is different for each random variate
*
*/
	thrust::device_vector<float> rand1(samples_needed); 
	float *rand_ptr1 = raw_pointer_cast(&rand1[0]);
	hiprandGenerateUniform(gen, rand_ptr1, samples_needed);

	thrust::fill(random_variates.begin(), random_variates.end(), 0);

	thrust::for_each( thrust::make_zip_iterator(thrust::make_tuple(rand1.begin(),probabilities.begin(), random_variates.begin() )),
			thrust::make_zip_iterator(thrust::make_tuple(rand1.begin() + samples_needed, probabilities.begin() + samples_needed, random_variates.begin() + samples_needed)),
			bernoulli_rv_different_parameters());		
	}
