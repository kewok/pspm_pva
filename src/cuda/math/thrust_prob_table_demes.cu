#include <math/thrust_prob_table_demes.h>
#include <util/footimer2.h>
#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/distance.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/gather.h>

// make sure the individual's uniform random number falls in the interval corresponding to their current deme
struct adjust_randoms_functor
	{
	float *popRanges_in_cumulative;

	adjust_randoms_functor(float *pop_range) : popRanges_in_cumulative(pop_range)
	{};
	
	/*
		Elements in the tuple.
		----------------------
		0: the individual's random number
		1: the individual's deme
	*/
	template <typename tuple>
	__host__ __device__
	void operator()(tuple t) {
			int this_inds_deme = thrust::get<1>(t);
			if (this_inds_deme != 0)
				{
				float interval = popRanges_in_cumulative[this_inds_deme] - popRanges_in_cumulative[this_inds_deme - 1];
				thrust::get<0>(t) = thrust::get<0>(t)*interval +  popRanges_in_cumulative[this_inds_deme - 1];
				}

	 // For the case of the individuals in deme zero
			else
				{
				float interval = popRanges_in_cumulative[this_inds_deme];
				thrust::get<0>(t) = thrust::get<0>(t)*interval;
				}
		}
	};

struct adjust_randoms_functor_double
	{
	double *popRanges_in_cumulative;

	adjust_randoms_functor_double(double *pop_range) : popRanges_in_cumulative(pop_range)
	{};
	
	/*
		Elements in the tuple.
		----------------------
		0: the individual's random number
		1: the individual's deme
	*/
	template <typename tuple>
	__host__ __device__
	void operator()(tuple t) {
			int this_inds_deme = thrust::get<1>(t);
			if (this_inds_deme != 0)
				{
				double interval = popRanges_in_cumulative[this_inds_deme] - popRanges_in_cumulative[this_inds_deme - 1];
				thrust::get<0>(t) = thrust::get<0>(t)*interval +  popRanges_in_cumulative[this_inds_deme - 1];
				}

	 // For the case of the individuals in deme zero
			else
				{
				double interval = popRanges_in_cumulative[this_inds_deme];
				thrust::get<0>(t) = thrust::get<0>(t)*interval;
				}
		}
	};


void ThrustProbTable_demes::adjust_randoms(thrust::device_vector<float>::iterator uniform_begin, thrust::device_vector<float>::iterator uniform_end,
thrust::device_vector<int>::iterator deme_offsets_begin, thrust::device_vector<int>::iterator deme_offsets_end,
thrust::device_vector<int>::iterator inds_deme_begin, thrust::device_vector<int>::iterator inds_deme_end)
	{
	// Determine the number of demes.
	int n = thrust::distance(deme_offsets_begin, deme_offsets_end);

	// Figure out the bounds for each deme in terms of their values in the cumulative probability table
	thrust::device_vector<float> bounds(n);

	thrust::gather(deme_offsets_begin, deme_offsets_end,cumulative_prob.begin(),bounds.begin());
	
	thrust::device_vector<int> temp(n);
	thrust::copy(deme_offsets_begin, deme_offsets_end, temp.begin());

	float *cumul_prob_bounds = raw_pointer_cast(&bounds[0]);
	// Instantiate the random number adjuster
	adjust_randoms_functor adjuster(cumul_prob_bounds);

	// Adjust the random numbers to fall inside the correct interval
	thrust::for_each( thrust::make_zip_iterator(thrust::make_tuple(
				uniform_begin, 
				inds_deme_begin
				)),
			thrust::make_zip_iterator(thrust::make_tuple(
				uniform_end,
				inds_deme_end 
				)),
				adjuster
				);
	}



void ThrustProbTable_demes::adjust_randoms(thrust::device_vector<double>::iterator uniform_begin, thrust::device_vector<double>::iterator uniform_end,
thrust::device_vector<int>::iterator deme_offsets_begin, thrust::device_vector<int>::iterator deme_offsets_end,
thrust::device_vector<int>::iterator inds_deme_begin, thrust::device_vector<int>::iterator inds_deme_end)
	{
	// Determine the number of demes.
	int n = thrust::distance(deme_offsets_begin, deme_offsets_end);

	// Figure out the bounds for each deme in terms of their values in the cumulative probability table
	thrust::device_vector<double> bounds(n);

	thrust::gather(deme_offsets_begin, deme_offsets_end,cumulative_prob.begin(),bounds.begin());
	
	thrust::device_vector<int> temp(n);
	thrust::copy(deme_offsets_begin, deme_offsets_end, temp.begin());

	double *cumul_prob_bounds = raw_pointer_cast(&bounds[0]);
	// Instantiate the random number adjuster
	adjust_randoms_functor_double adjuster(cumul_prob_bounds);

	// Adjust the random numbers to fall inside the correct interval
	thrust::for_each( thrust::make_zip_iterator(thrust::make_tuple(
				uniform_begin, 
				inds_deme_begin
				)),
			thrust::make_zip_iterator(thrust::make_tuple(
				uniform_end,
				inds_deme_end 
				)),
				adjuster
				);
	}

