#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <iostream>
#include <stdio.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/distance.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/replace.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/scan.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>
#include <thrust/gather.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/sort.h>
#include <thrust/remove.h>
#include <thrust/unique.h>
#include <thrust/set_operations.h>
#include <thrust/extrema.h>

#include <util/thrust_functors.h>
#include <math/thrust_probabilities.h>
#include <math/histogram.h>
/* 

Stolen from:
http://code.google.com/p/thrust/source/browse/examples/histogram.cu

*/

// TODO: Add a case for handling what to do when data are of size zero.

void calculate_histogram(thrust::device_vector<int> &data,
			 thrust::device_vector<int> &counts,
			 int counts_size)
	{
	counts.resize( counts_size );
	
	int data_size = data.size();
	thrust::counting_iterator<int> search_begin(0);
	thrust::device_vector<int> temp_data( data_size );

	thrust::copy(data.begin(), data.end(), temp_data.begin());

	thrust::sort(temp_data.begin(), temp_data.end());

	thrust::upper_bound(temp_data.begin(),  temp_data.end(), 
                      search_begin, search_begin + counts_size,
                      counts.begin());

	thrust::adjacent_difference(counts.begin(), counts.end(),
                              counts.begin());
	}


void calculate_histogram(thrust::device_vector<float> &data,
			 thrust::device_vector<int> &counts,
			 int counts_size)
	{
	counts.resize( counts_size );
	
	int data_size = data.size();
	thrust::counting_iterator<int> search_begin(0);
	thrust::device_vector<float> temp_data( data_size );

	float maxval=*(thrust::max_element(data.begin(), data.end()));
	float minval=*(thrust::min_element(data.begin(), data.end()));

	thrust::device_vector<float> maxes( data_size );
	thrust::device_vector<float> mins( data_size );
	thrust::device_vector<int> bin_nums( data_size );

	thrust::fill(maxes.begin(), maxes.end(), maxval);
	thrust::fill(mins.begin(), mins.end(), minval);
	thrust::fill(bin_nums.begin(), bin_nums.end(), counts_size);
	
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(data.begin(), mins.begin(), maxes.begin(), bin_nums.begin(), temp_data.begin())),
			 thrust::make_zip_iterator(thrust::make_tuple(data.end(), mins.end(), maxes.end(), bin_nums.end(), temp_data.end())),
			 adjust_histogram_data_values());

	thrust::sort(temp_data.begin(), temp_data.end());

	thrust::upper_bound(temp_data.begin(),  temp_data.end(), 
                      search_begin, search_begin + counts_size,
                      counts.begin());

	thrust::adjacent_difference(counts.begin(), counts.end(),
                              counts.begin());	
	}


/*****************
// To extract the counts for a subset of the histogram elements pertaining to specific values
// e.g., data=(0,0,0,1,1,2,2,2,2,3) with histogram counts = (3,2,4,1), extract only counts for element 1 and 3 so end up with histogram = (2,1)

// The variable counts_size refers to the number of values data can take on

Todo: Add an error catch that if counts_size < max(data), return error 
//
*****************/

void calculate_histogram_subset(thrust::device_vector<int> &data,
			 thrust::device_vector<int> &counts,
			 int counts_size,
			 thrust::device_vector<int> &subset_indices)
	{
	calculate_histogram(data, counts, counts_size);

	thrust::device_vector<int> final_counts( subset_indices.size() );

	thrust::gather(subset_indices.begin(), subset_indices.end(), counts.begin(), final_counts.begin());

	counts.resize( subset_indices.size() );

	thrust::copy( final_counts.begin(), final_counts.end(), counts.begin() );
	}

