#include <math/thrust_prob_table.h>

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/distance.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>

void ThrustProbTable::setup(thrust::device_vector<float>::iterator prob_begin, thrust::device_vector<float>::iterator prob_end)
	{
	int n = thrust::distance(prob_begin, prob_end);
	cumulative_prob.resize(n);
	
	float total = thrust::reduce(prob_begin, prob_end);
	thrust::device_vector<float> total_vec(n);
	thrust::fill(total_vec.begin(), total_vec.end(), total);
	
	thrust::device_vector<float> relative_prob(n);
	thrust::transform(prob_begin, prob_end, total_vec.begin(), relative_prob.begin(), thrust::divides<float>());
	
	thrust::inclusive_scan(relative_prob.begin(), relative_prob.end(), cumulative_prob.begin());
	}

void ThrustProbTable::draw(thrust::device_vector<float>::iterator uniform_begin, thrust::device_vector<float>::iterator uniform_end, thrust::device_vector<int>::iterator result)
	{
	thrust::lower_bound(cumulative_prob.begin(), cumulative_prob.end(), uniform_begin, uniform_end, result);
	}

void ThrustProbTable::setup(thrust::device_vector<double>::iterator prob_begin, thrust::device_vector<double>::iterator prob_end)
	{
	int n = thrust::distance(prob_begin, prob_end);
	cumulative_prob_double.resize(n);
	
	double total = thrust::reduce(prob_begin, prob_end);
	thrust::device_vector<double> total_vec(n);
	thrust::fill(total_vec.begin(), total_vec.end(), total);
	
	thrust::device_vector<double> relative_prob(n);
	thrust::transform(prob_begin, prob_end, total_vec.begin(), relative_prob.begin(), thrust::divides<double>());
	
	thrust::inclusive_scan(relative_prob.begin(), relative_prob.end(), cumulative_prob_double.begin());
	}

void ThrustProbTable::draw(thrust::device_vector<double>::iterator uniform_begin, thrust::device_vector<double>::iterator uniform_end, thrust::device_vector<int>::iterator result)
	{
	thrust::lower_bound(cumulative_prob_double.begin(), cumulative_prob_double.end(), uniform_begin, uniform_end, result);
	}
