#include <species/add_kids/neonates_class.h>

/* One problem with a separate neonates class is that all this stuff has to get copied back into example species. Performance cost seems minimal but more extensive profiling seems warranted. */

EggsNeonates::EggsNeonates(inds_stochastic *species, thrust::device_vector<int> &kids_per_mom) 
	{
	this->species = species;
	this->gen = species->gen;

	nphen = species->nphen;
	nloci = species->nloci;

	//Copy in the recombination map
	recomb_rate.resize(nloci);
	thrust::copy(species->demeParameters->GeneticArchitecture->recombination_rates.begin(), species->demeParameters->GeneticArchitecture->recombination_rates.begin() + nloci, recomb_rate.begin());

	previous_pop_size = species->size;

	Num_Demes = species->Num_Demes;
	Neonates_per_Deme.resize(Num_Demes);
	thrust::fill(Neonates_per_Deme.begin(), Neonates_per_Deme.end(), 0);

	Determine_Neonate_Population_Sizes(species->demeParameters, species->deme, kids_per_mom, species->deme_sizes, species->max_deme_sizes);

	kids_deme.resize(Total_Number_of_Neonates);

	amplify_sequence( Neonates_per_Deme, Num_Demes, kids_deme );

	// See to it that the allocated deme gets the neonates assigned.
	integrate_kids();

	mutation_magnitude.resize(Total_Number_of_Neonates);
	mutation_rate.resize(Total_Number_of_Neonates);
	}

void EggsNeonates::Determine_Neonate_Population_Sizes(DemeSettings *demeParameters,
						      thrust::device_vector<int> &everybodys_deme,
						      thrust::device_vector<int> &kids_per_mom,
						      thrust::device_vector<int> &current_deme_sizes,
						      thrust::device_vector<int> &maximum_deme_sizes)
	{
	/*
	* Because there are often more neonates than max_deme_sizes, this function culls the surplus neonates at random by determining the number of neonates each deme can contribute
	*/
	hipDeviceSynchronize();
	reduce_by_key_with_zeros(everybodys_deme, kids_per_mom, Neonates_per_Deme, previous_pop_size, Num_Demes); 

	// Make sure no population has more kids than there are spaces
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(Neonates_per_Deme.begin(), current_deme_sizes.begin(), maximum_deme_sizes.begin())),
			 thrust::make_zip_iterator(thrust::make_tuple(Neonates_per_Deme.end(), current_deme_sizes.end(), maximum_deme_sizes.end())),
	                 adjust_kids_functor());

	Total_Number_of_Neonates = thrust::reduce(Neonates_per_Deme.begin(), Neonates_per_Deme.end());
	}


void EggsNeonates::inherit_genotypes(thrust::device_vector<float> &probability_individuals_become_mothers,
				 thrust::device_vector<float> &probability_individuals_become_fathers)
	{
	get_maternally_derived_genotype(probability_individuals_become_mothers, species->mgenotype, species->fgenotype);
	get_paternally_derived_genotype(probability_individuals_become_fathers, species->mgenotype, species->fgenotype);
	mutate(species->fgenotype, species->mgenotype);
	}


void EggsNeonates::get_maternally_derived_genotype(thrust::device_vector<float> &probability_individuals_become_mothers,
					     thrust::device_vector<float> *&mgenotype,
					     thrust::device_vector<float> *&fgenotype)
	{
	/*
	* Calculate the offspring's genotypes at the maternally inherited loci. This function will determine who the mother is, and generate a haploid gamete from the mother that will be grafted on to the offspring.
	*/
	mating_ThrustProbTable_demes at;
	thrust::device_vector<int> mother_index(Total_Number_of_Neonates);
	thrust::device_vector<float> rand(Total_Number_of_Neonates);
	thrust::device_vector<int> parity(Total_Number_of_Neonates);
	float *rand_ptr = raw_pointer_cast(&rand[0]);
/*
	Feed reproductive probablity into the setup of the alias table.
	Draw from the alias table to determine mothers.
*/
	at.setup(probability_individuals_become_mothers.begin(), probability_individuals_become_mothers.begin() + previous_pop_size);
	hiprandGenerateUniform(gen, rand_ptr, Total_Number_of_Neonates);
	at.determine_key_offsets( Num_Demes, species->deme_sizes );
	at.adjust_randoms(rand.begin(), rand.end(), kids_deme.begin(), kids_deme.end());

	at.draw(rand.begin(), rand.end(), mother_index.begin());
	thrust::copy(mother_index.begin(), mother_index.end(), species->maternal_id.begin() + previous_pop_size);

//Initialize parity to zeroes
//Parity vector is used to keep track of where the recombination is happening.
	thrust::fill(parity.begin(), parity.end(), 0);


//Recombination for fgenotype
	for (int i = 0 ; i < nloci ; i++) 
		{
		hiprandGenerateUniform(gen, rand_ptr, Total_Number_of_Neonates);
        	recombine(rand, mother_index, parity, fgenotype, mgenotype, fgenotype[i], i);
		}
	}

void EggsNeonates::get_paternally_derived_genotype(thrust::device_vector<float> &probability_individuals_become_fathers,
					     thrust::device_vector<float> *&mgenotype,
					     thrust::device_vector<float> *&fgenotype)
	{
	/*
	* Calculate the offspring's genotypes at the paternally inherited loci. This function will determine who the mother is, and generate a haploid gamete from the mother that will be grafted on to the offspring.
	*/
	mating_ThrustProbTable_demes at;
	thrust::device_vector<int> father_index(Total_Number_of_Neonates);
	thrust::device_vector<float> rand(Total_Number_of_Neonates);
	thrust::device_vector<int> parity(Total_Number_of_Neonates);
	float *rand_ptr = raw_pointer_cast(&rand[0]);
	
	/*
	Directly use phenotype[1] as the reproductive probability.
	Reproductive probability = phenotype[1].
	Feed reproductive probablity into the setup of the alias table.
	Draw from the alias table to determine fathers.
	*/
	
	at.setup(probability_individuals_become_fathers.begin(), probability_individuals_become_fathers.begin() + previous_pop_size);
	hiprandGenerateUniform(gen, rand_ptr, Total_Number_of_Neonates);

	at.determine_key_offsets( Num_Demes, species->deme_sizes );	
	at.adjust_randoms(rand.begin(), rand.end(), kids_deme.begin(), kids_deme.end());

	at.draw(rand.begin(), rand.end(),father_index.begin());
	thrust::copy(father_index.begin(), father_index.end(), species->paternal_id.begin() + previous_pop_size);

	//Reset parity to zeroes
	thrust::fill(parity.begin(), parity.end(), 0);

	//Recombination for mgenotype
	for (int i = 0 ; i < nloci ; i++) 
		{
		hiprandGenerateUniform(gen, rand_ptr, Total_Number_of_Neonates);
		recombine(rand, father_index, parity, fgenotype, mgenotype, mgenotype[i], i);
		}
	}

void EggsNeonates::mutate(thrust::device_vector<float> *&mgenotype, thrust::device_vector<float> *&fgenotype)
	{
	/*
	* Determine, for each offspring's locus, whether there will be a mutation at that locus, and if so what the magnitude of that mutation will be and how that changes the offspring's allelic value. The current behavior assumes mutations are gaussian about the parental allelic value, and that the mutation parameters (mutation rate and sd of gaussian) vary by deme. Possible expansions include allowing the mutation rate itself to be an individual-specific phenotype, or alternative mutational models (e.g., point mutations that have a categorical rather than quantitative effect.
	*/
	thrust::identity<int> identity;

	// Note throughout we use 2*Total_Number_of_Neonates; this is for performance reasons. drawing gaussians can be expensive so we want to parallelize as much of this as we can.
	thrust::device_vector<float> mutation_size(2*Total_Number_of_Neonates); 

	thrust::device_vector<float> mutation_prob(2*Total_Number_of_Neonates);

	thrust::device_vector<int> mutation_event(2*Total_Number_of_Neonates);
	thrust::fill(mutation_event.begin(), mutation_event.begin() + 2*Total_Number_of_Neonates, 0);
	// no mutation at the original sex determining locus

	for (int i = 0 ; i < nloci ; i++) 
		{
		// assign, for each offspring, the expected magnitude of the mutation (sd) and the expected mutation rate according to the their deme. At some point the performance of this regime needs to be profiled, because there are a lot of steps involved here.

		thrust::gather(kids_deme.begin(), kids_deme.begin() + Total_Number_of_Neonates, species->demeParameters->GeneticArchitecture->get_mutation_magnitudes_ptr(i), mutation_magnitude.begin());

		thrust::device_vector<float> mutation_size(2*Total_Number_of_Neonates); 

		thrust::copy(mutation_magnitude.begin(), mutation_magnitude.begin() + Total_Number_of_Neonates, mutation_size.begin());
		thrust::copy(mutation_magnitude.begin(), mutation_magnitude.begin() + Total_Number_of_Neonates, mutation_size.begin() + Total_Number_of_Neonates);

		thrust::gather(kids_deme.begin(), kids_deme.begin() + Total_Number_of_Neonates, species->demeParameters->GeneticArchitecture->get_mutation_rates_ptr(i), mutation_rate.begin());

		thrust::copy(mutation_rate.begin(), mutation_rate.begin() + Total_Number_of_Neonates, mutation_prob.begin());
		thrust::copy(mutation_rate.begin(), mutation_rate.begin() + Total_Number_of_Neonates, mutation_prob.begin() + Total_Number_of_Neonates);

		thrust::device_vector<float> mutation(2*Total_Number_of_Neonates);

		// Determine whether mutations will occur
		draw_bernoulli_different_parameters(2*Total_Number_of_Neonates, mutation_prob, mutation_event, gen);
		// Assume the mutation is symmetric about the current allelic value:
		draw_gaussian_different_parameters(2*Total_Number_of_Neonates, (float) 0.0, mutation_size, mutation, gen);
	
		// If there is a mutation, add it to the offspring's 
		thrust::transform_if(mgenotype[i].begin() + previous_pop_size, mgenotype[i].begin()  + previous_pop_size + Total_Number_of_Neonates, mutation.begin(), mutation_event.begin(), mgenotype[i].begin() + previous_pop_size, thrust::plus<float>(), identity);
		thrust::transform_if(fgenotype[i].begin() + previous_pop_size, fgenotype[i].begin()  + previous_pop_size + Total_Number_of_Neonates, mutation.begin() + Total_Number_of_Neonates, mutation_event.begin() + Total_Number_of_Neonates, fgenotype[i].begin()  + previous_pop_size, thrust::plus<float>(), identity);
		}
	}


void EggsNeonates::recombine(thrust::device_vector<float> &rand,
			     thrust::device_vector<int> &parent,
			     thrust::device_vector<int> &parity,
			     thrust::device_vector<float> *&parents_fgenotype,
			     thrust::device_vector<float> *&parents_mgenotype,
			     thrust::device_vector<float> &kids_genotype,
			     int locus_ID)
	{
/*
* A method that implements recombination. Note that in many cases, the arguments for either (parents_fgenotype or parents_mgenotype) and kids_genotype will be the same vector. This function increments the kids_genotype to begin at the (previous_pop_size)th index. This should probably be refactored so that there will be a kids_mgenotype and kids_fgenotype vector that is local to the neonates class, and that later gets copied into the inds class, i.e. something like: 
\code{.cpp}
	thrust::copy(kids_fgenotype[i].begin(), kids_fgenotype[i].end(), fgenotype[i].begin() + size);
\endcode
*/
	//Set up recombination functor.
	float *fgenotype_ptr = raw_pointer_cast(&parents_fgenotype[locus_ID][0]);
	float *mgenotype_ptr = raw_pointer_cast(&parents_mgenotype[locus_ID][0]);
	recombination_functor rfunc(fgenotype_ptr, mgenotype_ptr, recomb_rate[locus_ID]);
	
	//Perform recombination with arbitrary transform.
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(parent.begin(), rand.begin(), parity.begin(), kids_genotype.begin() + previous_pop_size)),
			 thrust::make_zip_iterator(thrust::make_tuple(parent.begin() + Total_Number_of_Neonates, rand.begin() + Total_Number_of_Neonates, parity.begin() + Total_Number_of_Neonates, kids_genotype.begin() +  previous_pop_size + Total_Number_of_Neonates)),
			 rfunc);
	}


void EggsNeonates::integrate_kids()
	{
	thrust::copy(kids_deme.begin(), kids_deme.begin() + Total_Number_of_Neonates, species->deme.begin() + previous_pop_size);
	thrust::sequence(species->id.begin() + previous_pop_size, species->id.begin() + previous_pop_size + Total_Number_of_Neonates, species->nextid);
	thrust::fill(species->status.begin() + previous_pop_size, species->status.begin() + previous_pop_size + Total_Number_of_Neonates, 1);
	thrust::fill(species->age.begin() + previous_pop_size, species->age.begin() + previous_pop_size + Total_Number_of_Neonates, 0);
	species->nextid += Total_Number_of_Neonates;
	species->size += Total_Number_of_Neonates;
	}
