#include <species/add_kids/assortative_mating_neonates_class.h>

/* One problem with a separate neonates class is that all this stuff has to get copied back into example species. It might just not be worth it depending on the performance cost. */

#define NLOCI 50

float recomb_array_assort[NLOCI] = {0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f,0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f,0.5f, 0.5f, 0.5f, 0.5f, 0.5f};


Assortative_mating_neonates::Assortative_mating_neonates(thrust::device_vector<int> &pair_populations, DemeSettings *subpopParameters, thrust::device_vector<int> &everybodys_deme, thrust::device_vector<int> &kids_per_mom,  thrust::device_vector<int> &current_deme_sizes, thrust::device_vector<int> &maximum_deme_sizes, int N_alive_inds,  int num_loci, int nPhen) : EggsNeonates(subpopParameters, everybodys_deme, kids_per_mom, current_deme_sizes, maximum_deme_sizes, N_alive_inds, num_loci, nPhen) 
	{

	pairs_per_deme.resize(Num_Subpopulations);
	
	// Determine how many pairs are in each subpopulation
	thrust::counting_iterator<int> search_begin(0);
	thrust::device_vector<int> temp_Subpop_sizes;

	temp_Subpop_sizes.resize(Num_Subpopulations);

	thrust::upper_bound(pair_populations.begin(), pair_populations.end(),
                      search_begin, search_begin + Num_Subpopulations,
                      temp_Subpop_sizes.begin());

	thrust::adjacent_difference(temp_Subpop_sizes.begin(), temp_Subpop_sizes.end(),
                              pairs_per_deme.begin());
		
	}

void Assortative_mating_neonates::inherit_genotypes_by_pair(thrust::device_vector<float> &probability_pair_becomes_parents,
				thrust::device_vector<int> &fathers_list,
				thrust::device_vector<int> &mothers_list,
				thrust::device_vector<float> *&fgenotype,
				thrust::device_vector<float> *&mgenotype,
				hiprandGenerator_t generator)
	{

	mothers_chosen.resize(Total_Number_of_Neonates);
	fathers_chosen.resize(Total_Number_of_Neonates);

	get_mating_pair(probability_pair_becomes_parents, fathers_list, mothers_list, generator);

	get_maternally_derived_genotype_deterministic(mothers_chosen, mgenotype, fgenotype, generator);

	get_paternally_derived_genotype_deterministic(fathers_chosen, mgenotype, fgenotype, generator);

	mutate(generator, fgenotype, mgenotype);
	}

void Assortative_mating_neonates::get_mating_pair(thrust::device_vector<float> &probability_pair_becomes_parents,
						   thrust::device_vector<int> &fathers_list,
						   thrust::device_vector<int> &mothers_list,
						   hiprandGenerator_t generator
						  )
	{


	mating_subpopThrustProbTable at;
	thrust::device_vector<int> pair_index(Total_Number_of_Neonates);
	thrust::device_vector<float> rand(Total_Number_of_Neonates);
	float *rand_ptr = raw_pointer_cast(&rand[0]);
/*
	Feed reproductive probablity into the setup of the alias table.
	Draw from the alias table to determine mothers.
*/
	at.setup(probability_pair_becomes_parents.begin(), probability_pair_becomes_parents.end());

	hiprandGenerateUniform(generator, rand_ptr, Total_Number_of_Neonates);
 
	at.determine_key_offsets( Num_Subpopulations, pairs_per_deme );
 
	at.adjust_randoms(rand.begin(), rand.end(), kids_pop.begin(), kids_pop.end());
 
	at.draw(rand.begin(), rand.end(), pair_index.begin());

 		
	thrust::gather(pair_index.begin(), pair_index.end(), fathers_list.begin(), fathers_chosen.begin());

 
	thrust::gather(pair_index.begin(), pair_index.end(), mothers_list.begin(), mothers_chosen.begin());

	}

/* use the functions blah_blah_deterministic() if the parents have already been chosen and you just need to copy genotypes */
	
void Assortative_mating_neonates::get_maternally_derived_genotype_deterministic(thrust::device_vector<int> &mother_index,
					     thrust::device_vector<float> *&mgenotype,
					     thrust::device_vector<float> *&fgenotype,
					     hiprandGenerator_t generator)
	{
	thrust::device_vector<float> rand(Total_Number_of_Neonates);
	float *rand_ptr = raw_pointer_cast(&rand[0]);
	hiprandGenerateUniform(generator, rand_ptr, Total_Number_of_Neonates);
	
	thrust::device_vector<int> parity(Total_Number_of_Neonates);
	thrust::fill(parity.begin(), parity.end(), 0);

	for (int i = 0 ; i < nloci ; i++) 
		{
		hiprandGenerateUniform(generator, rand_ptr, Total_Number_of_Neonates);
                recombine(rand, mother_index, parity, fgenotype, mgenotype, fgenotype[i], i);
		}
	}

void Assortative_mating_neonates::get_paternally_derived_genotype_deterministic(thrust::device_vector<int> &father_index,
					     thrust::device_vector<float> *&mgenotype,
					     thrust::device_vector<float> *&fgenotype,
					     hiprandGenerator_t generator)
	{
	thrust::device_vector<float> rand(Total_Number_of_Neonates);
	float *rand_ptr = raw_pointer_cast(&rand[0]);
	hiprandGenerateUniform(generator, rand_ptr, Total_Number_of_Neonates);
	
	//Reset parity to zeroes
	thrust::device_vector<int> parity(Total_Number_of_Neonates);
	thrust::fill(parity.begin(), parity.end(), 0);

	//Recombination for mgenotype
	for (int i = 0 ; i < nloci ; i++) 
		{
		hiprandGenerateUniform(generator, rand_ptr,Total_Number_of_Neonates);
		recombine(rand, father_index, parity, fgenotype, mgenotype, mgenotype[i], i);
		}
	}

void Assortative_mating_neonates::record_parents(thrust::device_vector<int> &maternal_id, thrust::device_vector<int> &paternal_id, thrust::device_vector<int> &ids)
	{
	thrust::gather(mothers_chosen.begin(), mothers_chosen.begin() + Total_Number_of_Neonates, ids.begin(), maternal_id.begin() + current_pop_size);
	thrust::gather(fathers_chosen.begin(), fathers_chosen.begin() + Total_Number_of_Neonates, ids.begin(), paternal_id.begin() + current_pop_size);
	}
