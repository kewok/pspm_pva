#include <species/add_kids/parents_class.h>
#include <species/add_kids/mating_kernel_functors.h>
#include <util/footimer2.h>

// since we are ignoring genetics and using an external hiprand generator here, this is OK:

Parents::Parents(inds_stochastic *species) 
	{
	this->phenotype = species->phenotype;

	// Get relevant values from inds_stochastic object
	size = species->size;
	this->Num_Demes = species->Num_Demes;
	gen = species->gen;
	this->demeParameters = species->demeParameters;

	deme.resize(size);
	thrust::copy(species->deme.begin(), species->deme.begin() + size, deme.begin());
	
	sex.resize(size);
	thrust::copy(species->sex.begin(), species->sex.begin() + size, sex.begin());

	// Configure the parents class accordingly
	will_reproduceF.resize(size);
	will_reproduceM.resize(size);

	probability_individual_becomes_female_parent.resize(size);
	probability_individual_becomes_male_parent.resize(size);

	kids_per_mom.resize(size);
	thrust::fill(kids_per_mom.begin(), kids_per_mom.begin() + size, 0);

	reproductive_males_per_deme.resize(Num_Demes); 
	thrust::fill(reproductive_males_per_deme.begin(), reproductive_males_per_deme.end(), 0);

	reproductive_females_per_deme.resize(Num_Demes); 
	thrust::fill(reproductive_females_per_deme.begin(), reproductive_females_per_deme.end(), 0);

	reproductive_potential_per_deme.resize(Num_Demes);
	thrust::fill(reproductive_potential_per_deme.begin(), reproductive_potential_per_deme.end(), 0);
	}

void Parents::setup_parents()
	{
	determine_parental_reproductive_potential();
	finalize_parental_reproductive_probabilities();
	}

void Parents::determine_parental_reproductive_potential()
	{
	/* these two steps determine_female_parent_eligibility and determine_male_parent_eligibility are almost certainly redundant and should be refactored */
	determine_female_parent_eligibility();
	determine_male_parent_eligibility();

	reduce_by_key_with_zeros(deme, will_reproduceM, reproductive_males_per_deme, size, Num_Demes); 

	female_fecundity();

	Potential_Number_of_Kids = thrust::reduce(kids_per_mom.begin(), kids_per_mom.end());

	if (Potential_Number_of_Kids < 0)
		{
		std::cout << "Potential number of kids shouldn't be negative" << std::endl;
		for (int i=0; i < kids_per_mom.size(); i++)
			std::cout << kids_per_mom[i] << std::endl;
		}
	}

void Parents::finalize_parental_reproductive_probabilities()
	{
	determine_probability_individual_becomes_female_parent();
	determine_probability_individual_becomes_male_parent();
	}

void Parents::determine_female_parent_eligibility()
	{
	// Default template that may be overriden by derived class 
	thrust::fill(will_reproduceF.begin(), will_reproduceF.begin() + size, 0);
	
	//Set up eligibility functor.
	parental_eligibility_functor parental_eligibility(0);
		
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(will_reproduceF.begin(), sex.begin())),
		 	thrust::make_zip_iterator(thrust::make_tuple(will_reproduceF.begin() + size, sex.begin() + size)),
			 parental_eligibility);	
	}

void Parents::determine_male_parent_eligibility()
	{
	// Default template that may be overriden by derived class 
	thrust::fill(will_reproduceM.begin(), will_reproduceM.begin() + size, 0);
	
	//Set up eligibility functor.
	parental_eligibility_functor parental_eligibility(1);
		
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(will_reproduceM.begin(), sex.begin())),
		 	thrust::make_zip_iterator(thrust::make_tuple(will_reproduceM.begin() + size, sex.begin() + size)),
			 parental_eligibility);	
	}

void Parents::female_fecundity()
	{
	int FECUNDITY_INDEX = demeParameters->species_specific_values["FECUNDITY_PHENOTYPE_INDEX"];
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(will_reproduceF.begin(), phenotype[FECUNDITY_INDEX].begin(), probability_individual_becomes_female_parent.begin())),
			 thrust::make_zip_iterator(thrust::make_tuple(will_reproduceF.begin() + size, phenotype[FECUNDITY_INDEX].begin() + size, probability_individual_becomes_female_parent.begin() + size)),
			 female_fecundity_functor());

	thrust::copy(probability_individual_becomes_female_parent.begin(), probability_individual_becomes_female_parent.end(), kids_per_mom.begin());
	}

void Parents::determine_probability_individual_becomes_female_parent()
	{
	/* Virtual void placeholders to be replaced by species-specific parents class*/
	}

void Parents::determine_probability_individual_becomes_male_parent()
	{
	/* Virtual void placeholder to be replaced by species-specific parents class*/
	}

