#include <species/update/updatebehavior.h>

void UpdateBehavior::determine_mortality(inds_stochastic *species)
	{
	/*
	* Code that actually changes the individual's vital state (dead=0, alive=1) during each time step. The value of the individual's probability of survival (stored in \code species->phenotype[MORTALITY_PHENOTYPE_INDEX] \endcode is set in an external function (e.g., something like \code update_mySpecies::determine_survivorship_probability() \endcode or something like that.).
	*/
	//Specify the individuals indices
	thrust::device_vector<int> individuals(species->size);
	thrust::sequence(individuals.begin(), individuals.begin() + species->size, 0);

	// Draw the random numbers
	thrust::device_vector<float> rand(species->size);
	// wrap the vector
	float *rand_ptr = raw_pointer_cast(&rand[0]);
	hiprandGenerateUniform(species->gen, rand_ptr, species->size);

	//Set up mortality.
	simulate_mortality mortality_functor(rand_ptr);

	//Perform mortality operation with for_each.
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(individuals.begin(), species->status.begin(), species->phenotype[species->MORTALITY_PHENOTYPE_INDEX].begin())),
			 thrust::make_zip_iterator(thrust::make_tuple(individuals.begin() + species->size, species->status.begin() + species->size, species->phenotype[species->MORTALITY_PHENOTYPE_INDEX].begin() + species->size)),
			 mortality_functor);
	}
