#include <species/inds.h>
#include <math/thrust_probabilities.h>

#include <util/rapidcsv/src/rapidcsv.h>

#include <hiprand.h>
#include <fstream>
#include <sstream>
#include <iostream>
#include <thrust/count.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/scatter.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>


inds::inds(int size_val, int maxsize_val, int num_demes, int species_ID_val) : size(size_val), maxsize(maxsize_val), Num_Demes(num_demes), nextid(size_val), species_ID(species_ID_val)
	{
	/*
	* A very limited initialization method which creates the data structures and performs a basic sanity check to make sure that the maximum and starting numbers of individuals are biologically meaningful.
	*/
	//Sanity check.
	if (size_val < 0 || maxsize_val < 0) {
		std::cerr << "Population size must be non-negative!" << std::endl;
		exit(1);
	}
	if (size_val > maxsize_val) {
		std::cerr << "Maximum size must be greater or equal to initial population size!" << std::endl;
		exit(1);
	}
	deme_sizes.resize(Num_Demes);
	max_deme_sizes.resize(Num_Demes);

	demeParameters = new DemeSettings("deme_config.txt", species_ID);

	if (demeParameters->check_number_of_demes() < Num_Demes)
		{
		std::cout << "The number of demes specified in the simulation_conf file cannot exceed the number specified in your deme_config.txt file; Please fix this before proceeding. " << std::endl;
		exit(1);
		}

	nloci = (int) demeParameters->GeneticArchitecture->Number_of_Loci;
	nphen = (int) demeParameters->GeneticArchitecture->Number_of_Phenotypes;

	initialize_individuals(nloci, nphen);
	//Set maxsize.
	setMaxSize(maxsize);

	demeCalculations();
	//Fill in ID, STATUS, and DEME.
	thrust::sequence(id.begin(), id.begin() + size);
	thrust::fill(status.begin(), status.begin() + size, 1);
	}

void inds::initialize_individuals(int nloci, int nphen)
	{
	//Allocate gene and phen data vectors.
	fgenotype = new thrust::device_vector<float>[nloci];
	mgenotype = new thrust::device_vector<float>[nloci];
	phenotype = new thrust::device_vector<float>[nphen];
	}

void inds::initialize_from_CSV(const char *filename)
	{
	rapidcsv::Document csv(filename);

	// Check that MaxSize is satisfied and exit if it isn't. 
	size = csv.GetRowCount();
	if (size > maxsize)
		{
		std::cerr << "Maximum size must be greater or equal to initial population size!" << std::endl;
		exit(1);
		}

	std::vector<int> id_in = csv.GetColumn<int>("Id");
	thrust::copy(id_in.begin(), id_in.end(), id.begin());

	std::vector<int> status_in = csv.GetColumn<int>("Status");
	thrust::copy(status_in.begin(), status_in.end(), status.begin());

	std::vector<int> sex_in = csv.GetColumn<int>("Sex");
	thrust::copy(sex_in.begin(), sex_in.end(), sex.begin());

	std::vector<int> age_in = csv.GetColumn<int>("Age");
	thrust::copy(age_in.begin(), age_in.end(), age.begin());

	std::vector<int> deme_in = csv.GetColumn<int>("Deme");
	thrust::copy(deme_in.begin(), deme_in.end(), deme.begin());
	
	std::string mgen_in = "mgene";
	std::string fgen_in = "fgene";

	for (int i=0; i < nloci; i++)
		{
		std::stringstream tempm;
		tempm << mgen_in << i ;
		std::stringstream tempf;
		tempf << fgen_in << i ;
	
		std::vector<float> mgenotype_in = csv.GetColumn<float>(tempm.str());
		std::vector<float> fgenotype_in = csv.GetColumn<float>(tempf.str());
		thrust::copy(mgenotype_in.begin(), mgenotype_in.end(), mgenotype[i].begin());
		thrust::copy(fgenotype_in.begin(), fgenotype_in.end(), fgenotype[i].begin());
		}

	std::string phen_in = "phen";
	for (int i=0; i < nphen; i++)
		{
		std::stringstream tempPh;
		tempPh << phen_in << i;
	
		std::vector<float> phenotype_in = csv.GetColumn<float>(tempPh.str());
		thrust::copy(phenotype_in.begin(), phenotype_in.end(), phenotype[i].begin());
		}

	// Optionally import parental IDs
	std::vector<std::string> columnNames = csv.GetColumnNames();
	bool maternal_ID_exists = (std::find(columnNames.begin(), columnNames.end(), "maternal_ID") != columnNames.end());
	if (maternal_ID_exists)
		{
		std::vector<int> maternal_id_in = csv.GetColumn<int>("maternal_ID");
		thrust::copy(maternal_id_in.begin(), maternal_id_in.end(), maternal_id.begin());
		}

	bool paternal_ID_exists = (std::find(columnNames.begin(), columnNames.end(), "paternal_ID") != columnNames.end());
	if (paternal_ID_exists)
		{
		std::vector<int> paternal_id_in = csv.GetColumn<int>("paternal_ID");
		thrust::copy(paternal_id_in.begin(), paternal_id_in.end(), paternal_id.begin());
		}
	}

inds::~inds()
	{
	/*
	* The destructor to clear GPU RAM; . Note only the thrust vectors of arrays are destroyed.
	*/
	delete[] fgenotype;
	delete[] mgenotype;
	delete[] phenotype;
	}

void inds::exportCsv()
	{
	std::ostringstream stream;
	stream << "species_" << species_ID << ".csv";
	std::string result = stream.str();
	const char *mySpeciesTitle = result.c_str();
	exportCsv(mySpeciesTitle);
	}

void inds::exportCsv(int timestep)
	{
	std::ostringstream stream;
	stream << "species_" << species_ID << "_" << timestep << ".csv";
	std::string result = stream.str();
	const char *mySpeciesTitle = result.c_str();
	exportCsv(mySpeciesTitle);
	}

void inds::exportCsv(const char *filename)
	{
	/*
	* Migrate all species-level data stored on the GPU ram to a CSV file on the main drive at any arbirary point in time.
	*/
	std::ofstream file;
	file.open(filename);
	
	//Output size
	//file << "Size,NLoci,NPhen,Seed" << std::endl;
	//file << size << "," << nloci << "," << nphen << "," << seed << std::endl << std::endl;
	
	//Output headers
	file << "Index,Id,Status,Sex,Age,Deme,";
	for (int i = 0 ; i < nloci ; i++) {
		file << "fgene" << i << ",";
	}
	for (int i = 0 ; i < nloci ; i++) {
		file << "mgene" << i << ",";
	}

	for (int i = 0 ; i < nphen ; i++) {
		if (i < nphen - 1)
			file << "phen" << i << ",";
		else
			file << "phen" << i << std::endl;
	}
	
	//Output n individuals
	for (int i = 0 ; i < size ; i++) {
		file << i << "," << id[i] << "," << status[i] << "," << sex[i] << "," << age[i] << "," << deme[i] << ",";
		for (int j = 0 ; j < nloci ; j++) {
			file << fgenotype[j][i] << ",";
		}
		for (int j = 0 ; j < nloci ; j++) {
			file << mgenotype[j][i] << ",";
		}
	for (int j = 0 ; j < nphen ; j++) {
		if (j < nphen - 1)
			file << phenotype[j][i] << ",";
		else
			file << phenotype[j][i] << std::endl;
		}
	}
	
	file.close();
	}

void inds::exportCsv(const char *filename, int timestep)
	{
	/*
	* An overloaded version of the export CSV file which prints parental ids of the individuals, as well as the time step yr/in which their data are recorded to the CSV file.
	*/
	std::ofstream file;
	file.open(filename,std::ios_base::app);
	
	//Output size
	//file << "Size,NLoci,NPhen,Seed" << std::endl;
	//file << size << "," << nloci << "," << nphen << "," << seed << std::endl << std::endl;
	
	//Output headers
	if ((timestep==0))
		{
		file << "Time_step, Index,Id,Status,Sex,Age,Deme,maternal_ID,paternal_ID,";
		for (int i = 0 ; i < nloci ; i++) {
			file << "fgene" << i << ",";
		}
		for (int i = 0 ; i < nloci ; i++) {
			file << "mgene" << i << ",";
		}
	
		for (int i = 0 ; i < nphen ; i++) {
			if (i < nphen - 1)
				file << "phen" << i << ",";
			else
				file << "phen" << i << std::endl;
		}
		}
	
	//Output n individuals
	for (int i = 0 ; i < size ; i++) {
		file << timestep << "," << i << "," << id[i] << "," << status[i] << "," << sex[i] << "," << age[i] << "," << deme[i] << "," << maternal_id[i] << "," << paternal_id[i] << ",";
		for (int j = 0 ; j < nloci ; j++) {
			file << fgenotype[j][i] << ",";
		}
		for (int j = 0 ; j < nloci ; j++) {
			file << mgenotype[j][i] << ",";
		}
	
		for (int j = 0 ; j < nphen ; j++) {
			if (j < nphen - 1)
				file << phenotype[j][i] << ",";
			else
				file << phenotype[j][i] << std::endl;
		}
	}
	
	file.close();
	}

void inds::exportCsv(const char *filename, int timestep1, int timestep2)
	{
	/*
	*Output file stream for each time-step timestep within yr which is incremented in main.cpp
	*/
	std::ofstream file;
	file.open(filename,std::ios_base::app);
	
	//Output size
	//file << "Size,NLoci,NPhen,Seed" << std::endl;
	//file << size << "," << nloci << "," << nphen << "," << seed << std::endl << std::endl;
	
	//Output headers
	if ((timestep1==0) && (timestep2 == 0))
		{
		file << "Time_Step1,Time_Step2,Index,Id,Status,Sex,Age,Deme,maternal_ID,paternal_ID";
	/*	for (int i = 0 ; i < nloci ; i++) {
			file << "fgene" << i << ",";
		}
		for (int i = 0 ; i < nloci ; i++) {
			file << "mgene" << i << ",";
		}
	*/
		for (int i = 0 ; i < nphen ; i++) {
			if (i < nphen - 1)
				file << "phen" << i << ",";
			else
				file << "phen" << i << std::endl;
			}
		}
	
	//Output n individuals
	for (int i = 0 ; i < size ; i++) {
		file << timestep1 << "," << timestep2 << "," << i << "," << id[i] << "," << status[i] << "," << sex[i] << "," << age[i] << "," << deme[i] << "," << maternal_id[i] << "," << paternal_id[i] << ",";
	/*	for (int j = 0 ; j < nloci ; j++) {
			file << fgenotype[j][i] << ",";
		}
		for (int j = 0 ; j < nloci ; j++) {
			file << mgenotype[j][i] << ",";
		}
	*/
	for (int j = 0 ; j < nphen ; j++) {
		if (j < nphen - 1)
			file << phenotype[j][i] << ",";
		else
			file << phenotype[j][i] << std::endl;
		}
	}
	
	file.close();
	}

void inds::removeDead()
	{
	/*
	* A function to reorganize your data points so that inds (0, 1, 2, ..., size) consist only of individuals whose vital status = 1.
	* This method rearranges the data structures of inds so that all the data points represented in each data structure from 0 to number_of_individuals represent individuals that are alive. The data points past number_of_individuals are garbage. All operations on inds or its derived classes should, therefore, operate only on number_of_individuals data points. As a rule, number_of_individuals <= max_number_of_individuals; when the two are equal, the data from dead individuals should simply be overwritten until number_of_individuals < max_number_of_individuals again.

	* Ideally this would involve stream compaction rather than relegating the dead individuals to occupy empty spaces, but some preliminary experiments suggested stream compaction entails substantial performance costs in thrust compared to the approach using sorting and gathering. This solution is subject to change; should performance improvements in Thrust or CUDA allow it, we will return to stream compaction routines here instead.
	*
	*/
	//Keys to flag those to be removed.
	thrust::device_vector<int> keys(size);
	
	//Setup keys. 1 = to be removed. 0 = not removed.
	thrust::transform(status.begin(), status.begin() + size, keys.begin(), thrust::logical_not<int>());

	reassign_dead_deme_functor give_deads_deme(Num_Demes);

	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(status.begin(), deme.begin())),
        		 thrust::make_zip_iterator(thrust::make_tuple(status.begin() + size, deme.begin() + size)),
			give_deads_deme);

	//Count number of alive individuals.
	int num_alive = thrust::count(status.begin(), status.begin() + size, 1);
	
	// Perform the sort based on deme with max_demes + 1 at the bottom
	// Then perform gather
	
	thrust::device_vector<int> map(size);
	thrust::copy(deme.begin(), deme.begin() + size, keys.begin());
	thrust::sequence(map.begin(), map.begin() + size);
	thrust::stable_sort_by_key(keys.begin(), keys.begin() + size, map.begin(), thrust::less<int>());

	thrust::device_vector<int> new_vals(size);
	thrust::device_vector<float> new_vals_float(size);

	thrust::gather(map.begin(), map.begin() + size, id.begin(), new_vals.begin());
	thrust::copy(new_vals.begin(), new_vals.begin() + size, id.begin());

	thrust::gather(map.begin(), map.begin() + size, age.begin(), new_vals.begin());
	thrust::copy(new_vals.begin(), new_vals.begin() + size, age.begin());


	thrust::gather(map.begin(), map.begin() + size, sex.begin(), new_vals.begin());
	thrust::copy(new_vals.begin(), new_vals.begin() + size, sex.begin());
	
	thrust::gather(map.begin(), map.begin() + size, status.begin(), new_vals.begin());
	thrust::copy(new_vals.begin(), new_vals.begin() + size, status.begin());

	thrust::gather(map.begin(), map.begin() + size, maternal_id.begin(), new_vals.begin());
	thrust::copy(new_vals.begin(), new_vals.begin() + size, maternal_id.begin());

	thrust::gather(map.begin(), map.begin() + size, paternal_id.begin(), new_vals.begin());
	thrust::copy(new_vals.begin(), new_vals.begin() + size, paternal_id.begin());

	thrust::gather(map.begin(), map.begin() + size, deme.begin(), new_vals.begin());
	thrust::copy(new_vals.begin(), new_vals.begin() + size, deme.begin());

	for (int i = 0 ; i < nloci ; i++) {
		thrust::gather(map.begin(), map.begin() + size, fgenotype[i].begin(), new_vals_float.begin());
		thrust::copy(new_vals_float.begin(), new_vals_float.begin() + size, fgenotype[i].begin());
		thrust::gather(map.begin(), map.begin() + size, mgenotype[i].begin(), new_vals_float.begin());
		thrust::copy(new_vals_float.begin(), new_vals_float.begin() + size, mgenotype[i].begin());
		}

	for (int i = 0 ; i < nphen ; i++) {
		thrust::gather(map.begin(), map.begin() + size, phenotype[i].begin(), new_vals_float.begin());
		thrust::copy(new_vals_float.begin(), new_vals_float.begin() + size, phenotype[i].begin());
		}

	size = num_alive;
	// Recalculate the number of individuals per deme
	demeCalculations();
	}

void inds::setMaxSize(int n)
	{
	/*
	* Allocate memory RAM on the GPU device to be used in your inds class during sPEGG simulation. 
	*/
	if (n < 0) {
		std::cerr << "setMaxSize: size must be non-negative!" << std::endl;
		return;
	}
	
	id.resize(n);
	status.resize(n);
	sex.resize(n);
	age.resize(n);
	deme.resize(n);

	// Initialize vectors storing parental arrays:
	maternal_id.resize(n);
	thrust::fill(maternal_id.begin(), maternal_id.begin() + n, -1);

	paternal_id.resize(n);
	thrust::fill(paternal_id.begin(), paternal_id.begin() + n, -1);
	
	for (int i = 0 ; i < nloci ; i++) {
		fgenotype[i].resize(n);
		mgenotype[i].resize(n);
	}
	for (int i = 0 ; i < nphen ; i++) {
		phenotype[i].resize(n);
	}
	
	maxsize = n;
	}

void inds::sortByDeme()
	{
	/* 
	* Reorganize the individual-level data points as needed, according to the deme to which the individual belongs.  As in \link removeDead() 
the data points past number_of_individuals are garbage. Note that the vital status for all individuals in sort_by_deme is assumed to be 1.
	*
	*/
	//Count number of alive individuals.
	int num_alive = thrust::count(status.begin(), status.begin() + size, 1);

	//Keys
	thrust::device_vector<int> keys(num_alive);
	
	/*
		For each vector,
		set keys to the deme vector.
		Then use the keys to sort.
	*/

	// Relocate the values according to the map generated by sort
	thrust::device_vector<int> map(num_alive);
	thrust::copy(deme.begin(), deme.begin() + num_alive, keys.begin());
	thrust::sequence(map.begin(), map.begin() + num_alive);
	thrust::stable_sort_by_key(keys.begin(), keys.begin() + num_alive, map.begin());

	// Map now directs where individuals should go

	thrust::device_vector<int> new_vals(size);
	thrust::device_vector<float> new_vals_float(size);
	
	thrust::gather(map.begin(), map.begin() + num_alive, id.begin(), new_vals.begin());

	thrust::copy(new_vals.begin(), new_vals.begin() + num_alive, id.begin());
	thrust::gather(map.begin(), map.begin() + num_alive, age.begin(), new_vals.begin());
	thrust::copy(new_vals.begin(), new_vals.begin() + num_alive, age.begin());
	

	thrust::gather(map.begin(), map.begin() + num_alive, sex.begin(), new_vals.begin());
	thrust::copy(new_vals.begin(), new_vals.begin() + num_alive, sex.begin());
	
	thrust::gather(map.begin(), map.begin() + num_alive, status.begin(), new_vals.begin());
	thrust::copy(new_vals.begin(), new_vals.begin() + num_alive, status.begin());

	thrust::gather(map.begin(), map.begin() + num_alive, maternal_id.begin(), new_vals.begin());
	thrust::copy(new_vals.begin(), new_vals.begin() + num_alive, maternal_id.begin());

	thrust::gather(map.begin(), map.begin() + num_alive, paternal_id.begin(), new_vals.begin());
	thrust::copy(new_vals.begin(), new_vals.begin() + num_alive, paternal_id.begin());

	thrust::gather(map.begin(), map.begin() + num_alive, deme.begin(), new_vals.begin());
	thrust::copy(new_vals.begin(), new_vals.begin() + num_alive, deme.begin());

	for (int i = 0 ; i < nloci ; i++) 
		{
		thrust::gather(map.begin(), map.begin() + num_alive, fgenotype[i].begin(), new_vals_float.begin());
		thrust::copy(new_vals_float.begin(), new_vals_float.begin() + num_alive, fgenotype[i].begin());
		thrust::gather(map.begin(), map.begin() + num_alive, mgenotype[i].begin(), new_vals_float.begin());
		thrust::copy(new_vals_float.begin(), new_vals_float.begin() + num_alive, mgenotype[i].begin());
		}

	for (int i = 0 ; i < nphen ; i++) {
			thrust::gather(map.begin(), map.begin() + num_alive, phenotype[i].begin(), new_vals_float.begin());
			thrust::copy(new_vals_float.begin(), new_vals_float.begin() + num_alive, phenotype[i].begin());
		}
	}

void inds::demeCalculations()
	{
	/*
	* Determine how many individuals are in each deme. The results are stored in vector deme_sizes, which is accessed via get_deme_sizes(thrust::device_vector<int> &deme_sizevals) where the vector deme_sizevals has already been declared and instantiated.
	*/
	thrust::counting_iterator<int> search_begin(0);
	thrust::device_vector<int> temp_deme_sizes;

	temp_deme_sizes.resize(Num_Demes);

	thrust::upper_bound(deme.begin(), deme.begin() + size,
                      search_begin, search_begin + Num_Demes,
                      temp_deme_sizes.begin());

	thrust::adjacent_difference(temp_deme_sizes.begin(), temp_deme_sizes.end(),
                              deme_sizes.begin());
	}
