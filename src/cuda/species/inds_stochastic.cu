#include <species/inds_stochastic.h>

inds_stochastic::inds_stochastic(int size_val, int maxsize_val, int seed_val, int ndemes, int species_ID_val) : inds(size_val, maxsize_val, ndemes, species_ID_val)
	{
/*
*
* Initialize the hiprand generator gen based on the seed argument, using HIPRAND_RNG_PSEUDO_DEFAULT. Draw 100 random uniform variables and store them in the vector rand; then discard the contents of the rand vector. This is all done to initialize the CUDA random number generator object. Note the rand vector will then be deallocated once the prime_random_number_generator scope is ended. For reasons not entirely clear, this can't seem to be done inside an external function prime_random_number_generator
*
*/
	seed = seed_val;

	int size = 100;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, seed);

	//hiprand declarations
	thrust::device_vector<float> rand(size);
	float *rand_ptr = raw_pointer_cast(&rand[0]);
	hiprandGenerateUniform(gen, rand_ptr, size); // priming up the random number generator takes some time, get it done early.
	rand.clear();

	//Specify the indices among the phenotypes for the fitness components
	MORTALITY_PHENOTYPE_INDEX = (int) demeParameters->species_specific_values["MORTALITY_PHENOTYPE_INDEX"];
	FECUNDITY_PHENOTYPE_INDEX = (int) demeParameters->species_specific_values["FECUNDITY_PHENOTYPE_INDEX"];
	}

